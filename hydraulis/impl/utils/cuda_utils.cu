#include "hydraulis/impl/utils/cuda_utils.h"

namespace hydraulis {
namespace cuda {

namespace {
thread_local int current_device_id = -1;
} // namespace

#if CUDA_VERSION >= 12000
void CudaTryGetDevice(int* device_id) {
  *device_id = current_device_id;
}

void CudaSetDevice(int device_id) {
  // HT_LOG_INFO << "device id is " << device_id << " and old device id is " << current_device_id;
  if (current_device_id != device_id) {
    CUDA_CALL(hipSetDevice(device_id));
    current_device_id = device_id;
  }
}
#endif

NDArray to_int64_ndarray(const std::vector<int64_t>& vec,
                         DeviceIndex device_id) {
  auto ret = NDArray::empty({static_cast<int64_t>(vec.size())},
                            Device(kCUDA, device_id), kInt64, kBlockingStream);
  hydraulis::cuda::CUDADeviceGuard guard(device_id);
  CudaMemcpy(ret->raw_data_ptr(), vec.data(), vec.size() * sizeof(int64_t),
             hipMemcpyHostToDevice);
  return ret;
}

NDArray to_int64_ndarray(const int64_t* from, size_t n, DeviceIndex device_id) {
  auto ret = NDArray::empty({static_cast<int64_t>(n)}, Device(kCUDA, device_id),
                            kInt64, kBlockingStream);
  CudaMemcpy(ret->raw_data_ptr(), from, n * sizeof(int64_t),
             hipMemcpyHostToDevice);
  return ret;
}

NDArray to_byte_ndarray(const std::vector<uint8_t>& vec,
                        DeviceIndex device_id) {
  auto ret = NDArray::empty({static_cast<int64_t>(vec.size())},
                            Device(kCUDA, device_id), kByte, kBlockingStream);
  hydraulis::cuda::CUDADeviceGuard guard(device_id);
  CudaMemcpy(ret->raw_data_ptr(), vec.data(), vec.size() * sizeof(uint8_t),
             hipMemcpyHostToDevice);
  return ret;
}

NDArray to_byte_ndarray(const uint8_t* from, size_t n, DeviceIndex device_id) {
  auto ret = NDArray::empty({static_cast<int64_t>(n)}, Device(kCUDA, device_id),
                            kByte, kBlockingStream);
  CudaMemcpy(ret->raw_data_ptr(), from, n * sizeof(uint8_t),
             hipMemcpyHostToDevice);
  return ret;
}

} // namespace cuda
} // namespace hydraulis
