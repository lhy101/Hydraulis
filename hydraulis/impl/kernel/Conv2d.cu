#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/core/memory_pool.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/cuda/CUDADnn.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include <chrono>

namespace hydraulis {
namespace impl {

void Conv2dCuda(const NDArray& input_x, const NDArray& input_f, NDArray& output,
                const int padding_h, const int padding_w, const int stride_h,
                const int stride_w, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_x);
  HT_ASSERT_SAME_DEVICE(input_x, input_f);
  HT_ASSERT_SAME_DEVICE(input_x, output);

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hydraulis::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype = to_cudnn_DataType(input_x->dtype());

  size_t input_N = input_x->shape(0);
  size_t input_C = input_x->shape(1);
  size_t input_H = input_x->shape(2);
  size_t input_W = input_x->shape(3);

  size_t filter_N = input_f->shape(0);
  size_t filter_C = input_f->shape(1);
  size_t filter_H = input_f->shape(2);
  size_t filter_W = input_f->shape(3);

  size_t out_N = output->shape(0);
  size_t out_C = output->shape(1);
  size_t out_H = output->shape(2);
  size_t out_W = output->shape(3);

  // input
  hipdnnTensorDescriptor_t input_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                        datatype, input_N, input_C, input_H,
                                        input_W));

  // filter
  hipdnnFilterDescriptor_t filter_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, datatype,
                                        HIPDNN_TENSOR_NCHW, filter_N,
                                        filter_C, filter_H, filter_W));

  // convolution
  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
    HIPDNN_CROSS_CORRELATION, input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16 ? HIPDNN_DATA_FLOAT : datatype));
  if (input_x->dtype() == DataType::FLOAT16)
    CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
  // output
  hipdnnTensorDescriptor_t out_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, datatype, out_N, out_C, out_H, out_W));
  // algorithm
  hipdnnConvolutionFwdAlgo_t algo;
  size_t workspace_size = 0;
  NDArray workspace;

#if defined(CUDNN_MAJOR) && ((CUDNN_MAJOR >= 8))
  // workaround here
  // TODO: using hipdnnFindConvolutionForwardAlgorithm in CuDNN 8 instead
  int return_algo_cnt = HIPDNN_CONVOLUTION_FWD_ALGO_COUNT;
  hipdnnConvolutionFwdAlgoPerf_t
    perf_results[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
  CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(
    handle, input_desc, filter_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_ALGO_COUNT, &return_algo_cnt, perf_results));

  void* tmp_work_data = nullptr;
  bool flag = false;
  for (int i = 0; i < return_algo_cnt; ++i) {
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
      handle, input_desc, filter_desc, conv_desc, out_desc,
      perf_results[i].algo, &workspace_size));
    if (hipMalloc(&tmp_work_data, workspace_size) == hipSuccess) {
      algo = perf_results[i].algo;
      CudaFree(tmp_work_data);
      flag = true;
      break;
    }
  }
  HT_RUNTIME_ERROR_IF(!flag) << "Memory insufficient to create workspace";
#else
  CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
      handle, input_desc, filter_desc, conv_desc, out_desc, 
      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
#endif
  CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    handle, input_desc, filter_desc, conv_desc, out_desc, algo,
    &workspace_size));

  if (workspace_size != 0) {
    workspace = NDArray::empty({static_cast<int64_t>(workspace_size)},
                               input_x->device(), kInt8, stream.stream_index());
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_x->dtype(), spec_t, "Conv2dCuda", [&]() {
      void* workspace_ptr =
        workspace.is_defined() ? workspace->raw_data_ptr() : nullptr;

      spec_t alpha = 1.0f;
      spec_t beta = 0.0f;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnConvolutionForward(handle, &alpha_f, input_desc, input_x->data_ptr<spec_t>(),
                                           filter_desc, input_f->data_ptr<spec_t>(), conv_desc,
                                           algo, workspace_ptr, workspace_size, &beta_f,
                                           out_desc, output->data_ptr<spec_t>()));
      } else {
        CUDNN_CALL(hipdnnConvolutionForward(handle, &alpha, input_desc, input_x->data_ptr<spec_t>(),
                                           filter_desc, input_f->data_ptr<spec_t>(), conv_desc,
                                           algo, workspace_ptr, workspace_size, &beta,
                                           out_desc, output->data_ptr<spec_t>()));
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
      CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
      CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    });
  NDArray::MarkUsedBy({input_x, input_f, output, workspace}, stream);
  return;
}

void Conv2dGradientofFilterCuda(const NDArray& input_x,
                                const NDArray& gradient_y, NDArray& gradient_f,
                                const int padding_h, const int padding_w,
                                const int stride_h, const int stride_w,
                                const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_x);
  HT_ASSERT_SAME_DEVICE(input_x, gradient_y);
  HT_ASSERT_SAME_DEVICE(input_x, gradient_f);

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hydraulis::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype = to_cudnn_DataType(input_x->dtype());

  // input
  size_t input_N = input_x->shape(0);
  size_t input_C = input_x->shape(1);
  size_t input_H = input_x->shape(2);
  size_t input_W = input_x->shape(3);
  // dy
  size_t dy_N = gradient_y->shape(0);
  size_t dy_C = gradient_y->shape(1);
  size_t dy_H = gradient_y->shape(2);
  size_t dy_W = gradient_y->shape(3);
  // dw
  size_t df_N = gradient_f->shape(0);
  size_t df_C = gradient_f->shape(1);
  size_t df_H = gradient_f->shape(2);
  size_t df_W = gradient_f->shape(3);

  // input
  hipdnnTensorDescriptor_t input_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                        datatype, input_N, input_C, input_H,
                                        input_W));

  // dy
  hipdnnTensorDescriptor_t dy_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(dy_desc, HIPDNN_TENSOR_NCHW,
                                        datatype, dy_N, dy_C, dy_H, dy_W));

  // conv2d
  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
    HIPDNN_CROSS_CORRELATION, input_x->dtype() == DataType::FLOAT16 || 
    input_x->dtype() == DataType::BFLOAT16 ? HIPDNN_DATA_FLOAT : datatype));
  if (input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16)
    CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));

  // dw
  hipdnnFilterDescriptor_t df_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&df_desc));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(
    df_desc, datatype, HIPDNN_TENSOR_NCHW, df_N, df_C, df_H, df_W));
  // algo
  hipdnnConvolutionBwdFilterAlgo_t algo;
  size_t workspace_size = 0;
  NDArray workspace;

#if defined(CUDNN_MAJOR) && ((CUDNN_MAJOR >= 8))
  // TODO: using hipdnnFindConvolutionBackwardFilterAlgorithm in CuDNN 8
  // instead algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT;
  int return_algo_cnt = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT;
  hipdnnConvolutionBwdFilterAlgoPerf_t
    perf_results[HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT];
  CUDNN_CALL(cudnnGetConvolutionBackwardFilterAlgorithm_v7(
    handle, input_desc, dy_desc, conv_desc, df_desc,
    HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT, &return_algo_cnt,
    perf_results));

  void* tmp_work_data = nullptr;
  bool flag = false;
  for (int i = 0; i < return_algo_cnt; ++i) {
    CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
      handle, input_desc, dy_desc, conv_desc, df_desc, perf_results[i].algo,
      &workspace_size));
    if (hipMalloc(&tmp_work_data, workspace_size) == hipSuccess) {
      algo = perf_results[i].algo;
      CudaFree(tmp_work_data);
      flag = true;
      break;
    }
  }
  HT_RUNTIME_ERROR_IF(!flag) << "Memory insufficient to create workspace";
#else
  CUDNN_CALL(hipdnnGetConvolutionBackwardFilterAlgorithm(
      handle, input_desc, dy_desc, conv_desc, df_desc,
      HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algo));
#endif
  CUDNN_CALL(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
    handle, input_desc, dy_desc, conv_desc, df_desc, algo,
    &workspace_size));

  if (workspace_size != 0) {
    workspace = NDArray::empty({static_cast<int64_t>(workspace_size)},
                               input_x->device(), kInt8, stream.stream_index());
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_x->dtype(), spec_t, "Conv2dGradientofFilterCuda", [&]() {
      void* workspace_ptr =
        workspace.is_defined() ? workspace->raw_data_ptr() : nullptr;

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnConvolutionBackwardFilter(
          handle, &alpha_f, input_desc, input_x->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta_f, df_desc, gradient_f->data_ptr<spec_t>()));
      } else {
        CUDNN_CALL(hipdnnConvolutionBackwardFilter(
          handle, &alpha, input_desc, input_x->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta, df_desc, gradient_f->data_ptr<spec_t>()));
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
      CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
      CUDNN_CALL(hipdnnDestroyFilterDescriptor(df_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    });
  NDArray::MarkUsedBy({input_x, gradient_y, gradient_f, workspace}, stream);
}

void Conv2dGradientofDataCuda(const NDArray& input_f, const NDArray& gradient_y,
                              NDArray& gradient_x, const int padding_h,
                              const int padding_w, const int stride_h,
                              const int stride_w, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_f);
  HT_ASSERT_SAME_DEVICE(input_f, gradient_y);
  HT_ASSERT_SAME_DEVICE(input_f, gradient_x);

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hydraulis::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype = to_cudnn_DataType(input_f->dtype());

  // filter
  size_t filter_N = input_f->shape(0);
  size_t filter_C = input_f->shape(1);
  size_t filter_H = input_f->shape(2);
  size_t filter_W = input_f->shape(3);
  // dy
  size_t dy_N = gradient_y->shape(0);
  size_t dy_C = gradient_y->shape(1);
  size_t dy_H = gradient_y->shape(2);
  size_t dy_W = gradient_y->shape(3);
  // dx
  size_t dx_N = gradient_x->shape(0);
  size_t dx_C = gradient_x->shape(1);
  size_t dx_H = gradient_x->shape(2);
  size_t dx_W = gradient_x->shape(3);

  // filter
  hipdnnFilterDescriptor_t filter_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, datatype,
                                        HIPDNN_TENSOR_NCHW, filter_N,
                                        filter_C, filter_H, filter_W));
  // dy
  hipdnnTensorDescriptor_t dy_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&dy_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(dy_desc, HIPDNN_TENSOR_NCHW,
                                        datatype, dy_N, dy_C, dy_H, dy_W));
  // conv2d
  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
    HIPDNN_CROSS_CORRELATION, input_f->dtype() == DataType::FLOAT16 || input_f->dtype() == DataType::BFLOAT16 ? HIPDNN_DATA_FLOAT : datatype));
  if (input_f->dtype() == DataType::FLOAT16 || input_f->dtype() == DataType::BFLOAT16)
    CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));
  // dx
  hipdnnTensorDescriptor_t dx_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&dx_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(dx_desc, HIPDNN_TENSOR_NCHW,
                                        datatype, dx_N, dx_C, dx_H, dx_W));

  // algo
  hipdnnConvolutionBwdDataAlgo_t algo;
  size_t workspace_size = 0;
  NDArray workspace;

#if defined(CUDNN_MAJOR) && ((CUDNN_MAJOR >= 8))
  // TODO: using hipdnnFindConvolutionBackwardDataAlgorithm in CuDNN 8
  // instead
  int return_algo_cnt = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM;
  hipdnnConvolutionBwdDataAlgoPerf_t
    perf_results[HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM];
  CUDNN_CALL(cudnnGetConvolutionBackwardDataAlgorithm_v7(
    handle, filter_desc, dy_desc, conv_desc, dx_desc,
    HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM, &return_algo_cnt, perf_results));

  void* tmp_work_data = nullptr;
  bool flag = false;
  for (int i = 0; i < return_algo_cnt; ++i) {
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(
      handle, filter_desc, dy_desc, conv_desc, dx_desc,
      perf_results[i].algo, &workspace_size));
    if (hipMalloc(&tmp_work_data, workspace_size) == hipSuccess) {
      algo = perf_results[i].algo;
      CudaFree(tmp_work_data);
      flag = true;
      break;
    }
  }
  HT_RUNTIME_ERROR_IF(!flag) << "Memory insufficient to create workspace";
#else
  CUDNN_CALL(hipdnnGetConvolutionBackwardDataAlgorithm(
      handle, filter_desc, dy_desc, conv_desc, dx_desc,
      HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algo));
#endif
  CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(
    handle, filter_desc, dy_desc, conv_desc, dx_desc, algo,
    &workspace_size));

  if (workspace_size != 0) {
    workspace = NDArray::empty({static_cast<int64_t>(workspace_size)},
                               input_f->device(), kInt8, stream.stream_index());
  }

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_f->dtype(), spec_t, "Conv2dGradientofDataCuda", [&]() {
      void* workspace_ptr =
        workspace.is_defined() ? workspace->raw_data_ptr() : nullptr;

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (input_f->dtype() == DataType::FLOAT16 || input_f->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnConvolutionBackwardData(
          handle, &alpha_f, filter_desc, input_f->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta_f, dx_desc, gradient_x->data_ptr<spec_t>()));
      } else {
        CUDNN_CALL(hipdnnConvolutionBackwardData(
          handle, &alpha, filter_desc, input_f->data_ptr<spec_t>(), dy_desc, gradient_y->data_ptr<spec_t>(), 
          conv_desc, algo, workspace_ptr, workspace_size, &beta, dx_desc, gradient_x->data_ptr<spec_t>()));        
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(dy_desc));
      CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(dx_desc));
      CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
    });
  NDArray::MarkUsedBy({input_f, gradient_y, gradient_x, workspace}, stream);
}

template <typename spec_t>
__global__ void conv2d_add_bias_kernel(const spec_t* input, spec_t* output,
                                       size_t input_size, size_t output_size,
                                       size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t input_idx = (idx % input_size) / output_size;
  output[idx] = input[input_idx];
}

void Conv2dAddBiasCuda(const NDArray& input_x, const NDArray& input_f,
                       const NDArray& bias, NDArray& output,
                       const int padding_h, const int padding_w,
                       const int stride_h, const int stride_w,
                       const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input_x);
  HT_ASSERT_SAME_DEVICE(input_x, input_f);
  HT_ASSERT_SAME_DEVICE(input_x, bias);
  HT_ASSERT_SAME_DEVICE(input_x, output);

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hydraulis::impl::GetCudnnHandle(hip_stream.device_id());

  hipdnnDataType_t datatype = to_cudnn_DataType(input_x->dtype());

  size_t input_N = input_x->shape(0);
  size_t input_C = input_x->shape(1);
  size_t input_H = input_x->shape(2);
  size_t input_W = input_x->shape(3);

  size_t filter_N = input_f->shape(0);
  size_t filter_C = input_f->shape(1);
  size_t filter_H = input_f->shape(2);
  size_t filter_W = input_f->shape(3);

  size_t out_N = output->shape(0);
  size_t out_C = output->shape(1);
  size_t out_H = output->shape(2);
  size_t out_W = output->shape(3);

  // add bias
  size_t size = out_N * out_C * out_H * out_W;
  size_t bias_output_size = out_H * out_W;
  size_t bias_input_size = out_C * bias_output_size;

  // input
  hipdnnTensorDescriptor_t input_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                        datatype, input_N, input_C, input_H,
                                        input_W));
  // filter
  hipdnnFilterDescriptor_t filter_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, datatype,
                                        HIPDNN_TENSOR_NCHW, filter_N,
                                        filter_C, filter_H, filter_W));

  // convolution
  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
    conv_desc, padding_h, padding_w, stride_h, stride_w, 1, 1,
    HIPDNN_CROSS_CORRELATION, input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16? HIPDNN_DATA_FLOAT : datatype));
  if (input_x->dtype() == DataType::FLOAT16)
    CUDNN_CALL(hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH));

  // output
  hipdnnTensorDescriptor_t out_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(
    out_desc, HIPDNN_TENSOR_NCHW, datatype, out_N, out_C, out_H, out_W));
  // algorithm
  hipdnnConvolutionFwdAlgo_t algo;
  size_t workspace_size = 0;
  NDArray workspace;

#if defined(CUDNN_MAJOR) && ((CUDNN_MAJOR >= 8))
  // TODO: using hipdnnFindConvolutionForwardAlgorithm in CuDNN 8 instead
  int return_algo_cnt = HIPDNN_CONVOLUTION_FWD_ALGO_COUNT;
  hipdnnConvolutionFwdAlgoPerf_t
    perf_results[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
  CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(
    handle, input_desc, filter_desc, conv_desc, out_desc,
    HIPDNN_CONVOLUTION_FWD_ALGO_COUNT, &return_algo_cnt, perf_results));

  void* tmp_work_data = nullptr;
  bool flag = false;
  for (int i = 0; i < return_algo_cnt; ++i) {
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
      handle, input_desc, filter_desc, conv_desc, out_desc,
      perf_results[i].algo, &workspace_size));
    if (hipMalloc(&tmp_work_data, workspace_size) == hipSuccess) {
      algo = perf_results[i].algo;
      CudaFree(tmp_work_data);
      flag = true;
      break;
    }
  }
  HT_RUNTIME_ERROR_IF(!flag) << "Memory insufficient to create workspace";
#else
  CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
      handle, input_desc, filter_desc, conv_desc, out_desc,
      HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
#endif
  CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
    handle, input_desc, filter_desc, conv_desc, out_desc, algo,
    &workspace_size));

  if (workspace_size != 0) {
    workspace = NDArray::empty({static_cast<int64_t>(workspace_size)},
                               input_x->device(), kInt8, stream.stream_index());
  }

  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input_x->dtype(), spec_t, "Conv2dAddBiasCuda", [&]() {
      void* workspace_ptr =
        workspace.is_defined() ? workspace->raw_data_ptr() : nullptr;

      spec_t alpha = 1.0f;
      spec_t beta = 1.0f;

      float alpha_f = 1.0f;
      float beta_f = 1.0f;

      conv2d_add_bias_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        bias->data_ptr<spec_t>(), output->data_ptr<spec_t>(), bias_input_size,
        bias_output_size, size);

      if (input_x->dtype() == DataType::FLOAT16 || input_x->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnConvolutionForward(handle, &alpha_f, input_desc, input_x->data_ptr<spec_t>(),
                                          filter_desc, input_f->data_ptr<spec_t>(), conv_desc,
                                          algo, workspace_ptr, workspace_size, &beta_f,
                                          out_desc, output->data_ptr<spec_t>()));
      }  else {
        CUDNN_CALL(hipdnnConvolutionForward(handle, &alpha, input_desc, input_x->data_ptr<spec_t>(),
                                          filter_desc, input_f->data_ptr<spec_t>(), conv_desc,
                                          algo, workspace_ptr, workspace_size, &beta,
                                          out_desc, output->data_ptr<spec_t>()));
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
      CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
      CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
    });
  NDArray::MarkUsedBy({input_x, input_f, bias, output, workspace}, stream);
}

} // namespace impl
} // namespace hydraulis
