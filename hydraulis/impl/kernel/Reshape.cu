#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t>
__global__ void memory_copy_kernel(const spec_t* input, spec_t* output, size_t size,
                                   const OffsetCalculator* in_offset_calculator,
                                   const OffsetCalculator* out_offset_calculator) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  auto in_offset = in_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(idx);
  output[out_offset] = input[in_offset];
}

void ReshapeCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t input_size = input->numel();
  size_t size = output->numel();
  HT_ASSERT(input_size == size) << "input size and output size are different. "
                                << input_size << " " << size;
  if (input_size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ReshapeCuda", [&]() {
      memory_copy_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), size,
        in_offset_calculator, out_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
