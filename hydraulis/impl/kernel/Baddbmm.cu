#include "hydraulis/core/ndarray.h"
#include "hydraulis/core/stream.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/cuda/CUDABlas.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"

namespace hydraulis {
namespace impl {

void BaddbmmCuda(const NDArray& input, const NDArray& a, bool trans_a, const NDArray& b,
                 bool trans_b, float alpha, float beta, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(a);
  HT_ASSERT_SAME_DEVICE(a, input);
  HT_ASSERT_SAME_DEVICE(a, b);
  HT_ASSERT_SAME_DEVICE(a, output);
  HT_ASSERT_SAME_DTYPE(a, b);
  HT_ASSERT_SAME_DTYPE(a, output);

  hipblasHandle_t cublas_handle = GetCublasHandle(output->device().index());
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(output->device().index());

  int ndim = a->ndim();
  int m = output->shape(ndim - 1);
  int n = output->shape(ndim - 2);
  int k = trans_a ? a->shape(ndim - 2) : a->shape(ndim - 1);
  long long int strideA = a->shape(ndim - 2) * a->shape(ndim - 1);
  long long int strideB = b->shape(ndim - 2) * b->shape(ndim - 1);
  long long int strideC = output->shape(ndim - 2) * output->shape(ndim - 1);
  int batchCount = 1;
  for (int i = 0; i < ndim - 2; ++i) {
    HT_ASSERT(a->shape(i) == b->shape(i));
    HT_ASSERT(a->shape(i) == output->shape(i));
    batchCount *= a->shape(i);
  }

  size_t size = input->numel();

  HT_DISPATCH_FLOATING_TYPES(output->dtype(), spec_t, "BatchMatMul", [&]() {
    CudaMemcpyAsync(output->data_ptr<spec_t>(), input->data_ptr<spec_t>(), 
                    size * sizeof(spec_t), hipMemcpyDeviceToDevice, hip_stream);
    spec_t alpha1 = static_cast<spec_t>(alpha);
    spec_t beta1 = static_cast<spec_t>(beta);
    cublas_batch_gemm<spec_t>(
      cublas_handle, trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N,
      trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N, m, n, k, &alpha1,
      b->data_ptr<spec_t>(), trans_b ? k : m, strideB, a->data_ptr<spec_t>(),
      trans_a ? n : k, strideA, &beta1, output->data_ptr<spec_t>(), m, strideC,
      batchCount);
  });
  NDArray::MarkUsedBy({input, a, b, output}, stream);
}

} // namespace impl
} // namespace hydraulis
