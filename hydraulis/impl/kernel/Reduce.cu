#include "hydraulis/core/ndarray.h"
#include "hydraulis/core/memory_pool.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/cuda/CUDADnn.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"

namespace hydraulis {
namespace impl {

extern void ReduceSumCuda(const NDArray&, NDArray&, const int64_t*, int64_t,
                          const Stream&);
extern void ReduceMeanCuda(const NDArray&, NDArray&, const int64_t*, int64_t,
                           const Stream&);
extern void ReduceMaxCuda(const NDArray&, NDArray&, const int64_t*, int64_t,
                          const Stream&);
extern void ReduceMinCuda(const NDArray&, NDArray&, const int64_t*, int64_t,
                          const Stream&);
extern void ReduceProdCuda(const NDArray&, NDArray&, const int64_t*, int64_t,
                          const Stream&);

void CudnnReduceCuda(const NDArray& input, NDArray& output, const HTAxes& axes,
                     ReductionType red_type, const Stream& stream) {
  // TODO: Pack them up
  switch (red_type) {
    case kSUM:
      ReduceSumCuda(input, output, axes.data(), axes.size(), stream);
      break;
    case kMEAN:
      ReduceMeanCuda(input, output, axes.data(), axes.size(), stream);
      break;
    case kMAX:
      ReduceMaxCuda(input, output, axes.data(), axes.size(), stream);
      break;
    case kMIN:
      ReduceMinCuda(input, output, axes.data(), axes.size(), stream);
      break;
    case kPROD:
      ReduceProdCuda(input, output, axes.data(), axes.size(), stream);
      break;
    case kNONE:
      HT_NOT_IMPLEMENTED << "Reduction type cannot be none";
      __builtin_unreachable();
    default:
      HT_VALUE_ERROR << "Unknown reduction type: "
                     << static_cast<int32_t>(red_type);
      __builtin_unreachable();
  }
}

void ReduceCuda(const NDArray& input, NDArray& output, const HTAxes& axes,
                ReductionType red_type, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HTAxes parsed_axes = NDArrayMeta::ParseAxes(axes, input->ndim());
  CudnnReduceCuda(input, output, parsed_axes, red_type, stream);
}
} // namespace impl
} // namespace hydraulis
