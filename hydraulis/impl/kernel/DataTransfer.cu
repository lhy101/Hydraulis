#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"
#include "hydraulis/impl/kernel/Vectorized.cuh"

namespace hydraulis {
namespace impl {

bool require_temp_storage(const NDArray& from, const NDArray& to) {
  auto from_device = from->device();
  auto to_device = to->device();
  if (from_device == to_device) {
    return false;
  }
  bool same_dtype = from->dtype() == to->dtype();
  if (same_dtype && from->is_contiguous() && to->is_contiguous()
   || from_device.is_cuda() && to_device.is_cuda()) {
    return false;
  } else {
    return true;
  }
}

void transfer_device_to_device(const NDArray& from, NDArray& to, const Stream& stream) {
  size_t numel = from->numel();
  CUDAStream hip_stream(stream);

  bool memcpy_eligible = from->dtype() == to->dtype() &&
                         from->is_contiguous() && to->is_contiguous();
  auto from_device = from->device();
  auto to_device = to->device();
  void* to_ptr = to->raw_data_ptr();
  void* from_ptr = from->raw_data_ptr();

  if (memcpy_eligible) {
    size_t num_bytes = (from->dtype() == kFloat4 || from->dtype() == kNFloat4)
                       ? ((numel + 1) / 2) * DataType2Size(from->dtype())
                       : numel * DataType2Size(from->dtype());
    bool require_peer_memcpy = from->device().index() != to->device().index();
    
    if (to_ptr != from_ptr || from->device() != to->device()) {
      if (require_peer_memcpy) {
       // TODO: check that the stream belongs to source GPU as recommended by
       // https://www.nvidia.com/docs/IO/116711/sc11-multi-gpu.pdf.
        CudaMemcpyPeerAsync(to_ptr, to->device().index(),
                            from_ptr, from->device().index(),
                            num_bytes, hip_stream);
      } else {
        CudaMemcpyAsync(to_ptr, from_ptr, num_bytes, hipMemcpyDeviceToDevice,
                        hip_stream);
      }
    }
  } else {
    HT_DISPATCH_PAIRED_SIGNED_INTEGER_AND_FLOATING_TYPES(
      from->dtype(), to->dtype(), spec_a_t, spec_b_t, "DataTransferCuda",
      [&]() {
        launch_loop_kernel<spec_a_t, spec_b_t>(from, to, numel, stream,
                                               [] __device__ (spec_a_t x) -> spec_b_t {
                                                 return static_cast<spec_b_t>(x);
                                               });
      });
    NDArray::MarkUsedBy({from, to}, stream);
  }
}

void DataTransferCuda(const NDArray& from, NDArray& to, const Stream& stream) {
  HT_ASSERT_SAME_SHAPE(from, to);
  size_t numel = from->numel();
  if (numel == 0)
    return;

  if (require_temp_storage(from, to)) {
    auto from_dsize = DataType2Size(from->dtype());
    auto to_dsize = DataType2Size(to->dtype());
    NDArray from_contig, to_contig;
    if (to->device().is_cpu()) {
      from_contig = NDArray::contiguous(from, stream.stream_index());
      to_contig = NDArray::empty(to->shape(), from->device(), to->dtype()); 
      DataTransferCuda(from_contig, to_contig, stream);
      DataTransferCuda(to_contig, to, stream);
      NDArray::MarkUsedBy({from, to}, stream);
      return;
    } else if (from_dsize <= to_dsize) {
      auto from_converted = NDArray::to(from, from->device(), to->dtype(), stream.stream_index());
      from_contig = NDArray::contiguous(from_converted, stream.stream_index());
      to_contig = to->is_contiguous() ? to : NDArray::empty_like(to);
    } else {
      from_contig = NDArray::contiguous(from, stream.stream_index());
      to_contig = NDArray::empty(to->shape(), to->device(), from->dtype()); 
    }
    DataTransferCuda(from_contig, to_contig, stream);
    if (!to->is_contiguous() || to->dtype() != to_contig->dtype()) {
      HT_ASSERT(to_contig->device() == to->device());
      DataTransferCuda(to_contig, to, stream);
    }
    NDArray::MarkUsedBy({from, to}, stream);
    return;
  }

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());

  // Copy between GPUs
  if (from->device().is_cuda() && to->device().is_cuda()) {
    transfer_device_to_device(from, to, stream);
    NDArray::MarkUsedBy({from, to}, stream);
    return;
  }

  // Copy between CPU and GPU
  size_t num_bytes = (from->dtype() == kFloat4 || from->dtype() == kNFloat4) 
                     ? (numel + 1) / 2
                     : numel * DataType2Size(from->dtype());
  hipMemcpyKind kind;
  if (from->device().is_cuda() && to->device().is_cpu()) {
    kind = hipMemcpyDeviceToHost;
  } else if (from->device().is_cpu() && to->device().is_cuda()) {
    kind = hipMemcpyHostToDevice;
  } else {
    HT_RUNTIME_ERROR << "Cannot use DataTransferCuda to "
                     << "copy data between CPU tensors. "
                     << "Please use DataTransferCpu instead.";
  }
  CudaMemcpyAsync(to->raw_data_ptr(), from->raw_data_ptr(), num_bytes,
                  kind, hip_stream);
  NDArray::MarkUsedBy({from, to}, stream);
}

} // namespace impl
} // namespace hydraulis
