#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"
#include "hydraulis/impl/kernel/Vectorized.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t>
__global__ void check_finite_kernel(const spec_t* input, size_t size, float* output,
                                    const OffsetCalculator* in_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  auto in_offset = in_offset_calculator->get(idx);
  if (!isfinite(float(input[in_offset])))
    output[0] = 1.f;
  else 
    output[0] = 0.f;
}

void CheckFiniteCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "CheckFiniteCuda", [&]() {
      check_finite_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), size, output->data_ptr<float>(),
        in_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

template <typename spec_t>
__global__ void check_numeric_kernel(const spec_t* input, size_t size, float* output,
                                    const OffsetCalculator* in_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  auto in_offset = in_offset_calculator->get(idx);
  if (isnan(input[in_offset])) {
    output[0] = 1;
  }
  if (isinf(input[in_offset])) {
    output[input[in_offset] < spec_t(0.f) ? 1 : 2] = 1;
  }
}

void CheckNumericCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = input->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "CheckFiniteCuda", [&]() {
      launch_loop_kernel<spec_t>(output, size, stream,
                                 [=] __device__ (int /*idx*/) -> spec_t {
                                   return static_cast<spec_t>(0.f);
                                 });
      check_numeric_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), size, output->data_ptr<float>(),
        in_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
