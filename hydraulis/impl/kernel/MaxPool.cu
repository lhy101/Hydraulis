#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/cuda/CUDADnn.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"

namespace hydraulis {
namespace impl {

void MaxPoolCuda(const NDArray& input, const size_t kernel_H,
                 const size_t kernel_W, NDArray& output, const size_t padding,
                 const size_t stride, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hydraulis::impl::GetCudnnHandle(hip_stream.device_id());
  // input
  size_t input_N = input->shape(0);
  size_t input_C = input->shape(1);
  size_t input_H = input->shape(2);
  size_t input_W = input->shape(3);

  // output
  size_t output_H = output->shape(2);
  size_t output_W = output->shape(3);

  hipdnnDataType_t datatype = to_cudnn_DataType(input->dtype());

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "MaxPoolCuda", [&]() {
      const spec_t* input_data = (const spec_t*) input->data_ptr<spec_t>();
      spec_t* output_data = (spec_t*) output->data_ptr<spec_t>();
      // pooling descriptor
      hipdnnPoolingDescriptor_t maxpool_desc;
      CUDNN_CALL(hipdnnCreatePoolingDescriptor(&maxpool_desc));
      CUDNN_CALL(hipdnnSetPooling2dDescriptor(
        maxpool_desc, HIPDNN_POOLING_MAX_DETERMINISTIC, HIPDNN_PROPAGATE_NAN,
        kernel_H, kernel_W, padding, padding, stride, stride));

      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));

      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C,
                                            output_H, output_W));

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (input->dtype() == DataType::FLOAT16 || input->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnPoolingForward(handle, maxpool_desc, &alpha_f, input_desc,
                                      input_data, &beta_f, output_desc,
                                      output_data));
      } else {
        CUDNN_CALL(hipdnnPoolingForward(handle, maxpool_desc, &alpha, input_desc,
                                      input_data, &beta, output_desc,
                                      output_data));
      }

      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyPoolingDescriptor(maxpool_desc));
    });
  NDArray::MarkUsedBy({input, output}, stream);
}

void MaxPoolGradientCuda(const NDArray& output_Y, const NDArray& gradient_Y,
                         const NDArray& input_X, const size_t kernel_H,
                         const size_t kernel_W, NDArray& gradient_X,
                         const size_t padding, const size_t stride,
                         const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(output_Y);
  HT_ASSERT_SAME_DEVICE(output_Y, gradient_Y);
  HT_ASSERT_SAME_DEVICE(output_Y, input_X);
  HT_ASSERT_SAME_DEVICE(output_Y, gradient_X);

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hydraulis::impl::GetCudnnHandle(hip_stream.device_id());

  // input
  size_t input_N = input_X->shape(0);
  size_t input_C = input_X->shape(1);
  size_t input_H = input_X->shape(2);
  size_t input_W = input_X->shape(3);
  // output
  size_t output_H = output_Y->shape(2);
  size_t output_W = output_Y->shape(3);

  hipdnnDataType_t datatype = to_cudnn_DataType(output_Y->dtype());

  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    output_Y->dtype(), spec_t, "MaxPoolGradientCuda", [&]() {
      const spec_t* input_data = (const spec_t*) input_X->data_ptr<spec_t>();
      spec_t* gradient_x_data = (spec_t*) gradient_X->data_ptr<spec_t>();
      const spec_t* output_data = (const spec_t*) output_Y->data_ptr<spec_t>();
      const spec_t* gradient_Y_data =
        (const spec_t*) gradient_Y->data_ptr<spec_t>();
      // pooling descriptor
      hipdnnPoolingDescriptor_t maxpool_desc;
      CUDNN_CALL(hipdnnCreatePoolingDescriptor(&maxpool_desc));
      CUDNN_CALL(hipdnnSetPooling2dDescriptor(
        maxpool_desc, HIPDNN_POOLING_MAX_DETERMINISTIC, HIPDNN_PROPAGATE_NAN,
        kernel_H, kernel_W, padding, padding, stride, stride));

      // input descriptor
      hipdnnTensorDescriptor_t input_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C, input_H,
                                            input_W));

      // output descriptor
      hipdnnTensorDescriptor_t output_desc;
      CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_desc));
      CUDNN_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                            datatype, input_N, input_C,
                                            output_H, output_W));

      spec_t alpha = 1.0;
      spec_t beta = 0.0;

      float alpha_f = 1.0f;
      float beta_f = 0.0f;

      if (output_Y->dtype() == DataType::FLOAT16 || output_Y->dtype() == DataType::BFLOAT16) {
        CUDNN_CALL(hipdnnPoolingBackward(handle, maxpool_desc, &alpha_f, output_desc,
                                        output_data, output_desc, gradient_Y_data,
                                        input_desc, input_data, &beta_f, input_desc,
                                        gradient_x_data));
      } else {
        CUDNN_CALL(hipdnnPoolingBackward(handle, maxpool_desc, &alpha, output_desc,
                                        output_data, output_desc, gradient_Y_data,
                                        input_desc, input_data, &beta, input_desc,
                                        gradient_x_data));
      }
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_desc));
      CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_desc));
      CUDNN_CALL(hipdnnDestroyPoolingDescriptor(maxpool_desc));
    });
  NDArray::MarkUsedBy({output_Y, gradient_Y, input_X, gradient_X}, stream);
}

} // namespace impl
} // namespace hydraulis
