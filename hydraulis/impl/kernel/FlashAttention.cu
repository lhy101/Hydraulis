#include "hip/hip_runtime.h"
#include "cutlass/numeric_types.h"
#include "flash_attn/flash.h"
#include "flash_attn/static_switch.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/core/memory_pool.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/random/CPURandomState.h"
#include "hydraulis/impl/random/CUDARandomState.h"
#include "hydraulis/impl/cuda/CUDADnn.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/cuda_math.h"

namespace hydraulis {
namespace impl {

void set_params_fprop(Flash_fwd_params& params,
                      // sizes
                      const size_t b, const size_t seqlen_q,
                      const size_t seqlen_k, const size_t seqlen_q_rounded,
                      const size_t seqlen_k_rounded, const size_t h,
                      const size_t h_k, const size_t d, const size_t d_rounded,
                      // device pointers
                      const NDArray& q, const NDArray& k, const NDArray& v,
                      NDArray& out, void* cu_seqlens_q_d, void* cu_seqlens_k_d,
                      void* p_d, void* softmax_lse_d, float p_dropout,
                      float softmax_scale, bool is_causal) {
  // Reset the parameters
  memset(&params, 0, sizeof(params));

  params.is_bf16 = q->dtype() == kBFloat16;

  // Set the pointers and strides.
  params.q_ptr = q->raw_data_ptr();
  params.k_ptr = k->raw_data_ptr();
  params.v_ptr = v->raw_data_ptr();
  // All stride are in elements, not bytes.
  params.q_row_stride = q->stride(-3);
  params.k_row_stride = k->stride(-3);
  params.v_row_stride = v->stride(-3);
  params.q_head_stride = q->stride(-2);
  params.k_head_stride = k->stride(-2);
  params.v_head_stride = v->stride(-2);
  params.o_ptr = out->raw_data_ptr();
  params.o_row_stride = out->stride(-3);
  params.o_head_stride = out->stride(-2);

  if (cu_seqlens_q_d == nullptr) {
    params.q_batch_stride = q->stride(0);
    params.k_batch_stride = k->stride(0);
    params.v_batch_stride = v->stride(0);
    params.o_batch_stride = out->stride(0);
  }

  params.cu_seqlens_q = static_cast<int*>(cu_seqlens_q_d);
  params.cu_seqlens_k = static_cast<int*>(cu_seqlens_k_d);

  // P = softmax(QK^T)
  params.p_ptr = p_d;

  // Softmax sum
  params.softmax_lse_ptr = softmax_lse_d;

  // Set the dimensions.
  params.b = b;
  params.h = h;
  params.h_k = h_k;
  params.h_h_k_ratio = h / h_k;
  params.seqlen_q = seqlen_q;
  params.seqlen_k = seqlen_k;
  params.seqlen_q_rounded = seqlen_q_rounded;
  params.seqlen_k_rounded = seqlen_k_rounded;
  params.d = d;
  params.d_rounded = d_rounded;

  // Set the different scale values.
  params.scale_softmax = softmax_scale;
  params.scale_softmax_log2 = softmax_scale * M_LOG2E;

  // Set this to probability of keeping an element to simplify things.
  params.p_dropout = 1.f - p_dropout;
  // Convert p from float to int so we don't have to convert the random uint to
  // float to compare. [Minor] We want to round down since when we do the
  // comparison we use <= instead of < params.p_dropout_in_uint =
  // uint32_t(std::floor(params.p_dropout * 4294967295.0));
  // params.p_dropout_in_uint16_t = uint16_t(std::floor(params.p_dropout *
  // 65535.0));
  params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
  params.rp_dropout = 1.f / params.p_dropout;
  params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
  HT_ASSERT(p_dropout < 1.f) << "p_dropout > 1.";

  params.is_causal = is_causal;
  params.is_seqlens_k_cumulative = true;
}

void set_params_dgrad(Flash_bwd_params& params,
                      // sizes
                      const size_t b, const size_t seqlen_q,
                      const size_t seqlen_k, const size_t seqlen_q_rounded,
                      const size_t seqlen_k_rounded, const size_t h,
                      const size_t h_k, const size_t d, const size_t d_rounded,
                      // device pointers
                      const NDArray& q, const NDArray& k, const NDArray& v,
                      NDArray& out, NDArray& dout, NDArray& dq, NDArray& dk,
                      NDArray& dv, void* cu_seqlens_q_d, void* cu_seqlens_k_d,
                      void* dq_accum_d, void* dk_accum_d, void* dv_accum_d,
                      void* softmax_lse_d, void* dsoftmax_sum_d,
                      float p_dropout, float softmax_scale, bool is_causal) {
  set_params_fprop(params, b, seqlen_q, seqlen_k, seqlen_q_rounded,
                   seqlen_k_rounded, h, h_k, d, d_rounded, q, k, v, out,
                   cu_seqlens_q_d, cu_seqlens_k_d, nullptr, softmax_lse_d,
                   p_dropout, softmax_scale, is_causal);

  // Set the pointers and strides.
  params.do_ptr = dout->raw_data_ptr();
  params.do_row_stride = dout->stride(-3);
  params.do_head_stride = dout->stride(-2);
  params.dq_ptr = dq->raw_data_ptr();
  params.dk_ptr = dk->raw_data_ptr();
  params.dv_ptr = dv->raw_data_ptr();
  params.dq_row_stride = dq->stride(-3);
  params.dk_row_stride = dk->stride(-3);
  params.dv_row_stride = dv->stride(-3);
  params.dq_head_stride = dq->stride(-2);
  params.dk_head_stride = dk->stride(-2);
  params.dv_head_stride = dv->stride(-2);

  if (cu_seqlens_q_d == nullptr) {
    params.do_batch_stride = dout->stride(0);
    params.dq_batch_stride = dq->stride(0);
    params.dk_batch_stride = dk->stride(0);
    params.dv_batch_stride = dv->stride(0);
  }

  params.dq_accum_ptr = dq_accum_d;
  params.dk_accum_ptr = dk_accum_d;
  params.dv_accum_ptr = dv_accum_d;

  // Softmax sum
  params.dsoftmax_sum = dsoftmax_sum_d;
}

void run_mha_fwd(Flash_fwd_params& params, hipStream_t stream,
                 bool force_split_kernel = false) {
  FP16_SWITCH(!params.is_bf16, [&] {
    FWD_HEADDIM_SWITCH(params.d, [&] {
      if (params.num_splits <= 1 &&
          !force_split_kernel) { // If we don't set it num_splits == 0
        run_mha_fwd_<elem_type, kHeadDim>(params, stream);
      } else {
        run_mha_fwd_splitkv_dispatch<elem_type, kHeadDim>(params, stream);
      }
    });
  });
}

// Find the number of splits that maximizes the occupancy. For example, if we
// have batch * n_heads = 48 and we have 108 SMs, having 2 splits (efficiency =
// 0.89) is better than having 3 splits (efficiency = 0.67). However, we also
// don't want too many splits as that would incur more HBM reads/writes. So we
// find the best efficiency, then find the smallest number of splits that gets
// 85% of the best efficiency.
inline int num_splits_heuristic(int batch_nheads_mblocks, int num_SMs,
                                int num_n_blocks, int max_splits) {
  // If we have enough to almost fill the SMs, then just use 1 split
  if (batch_nheads_mblocks >= 0.8f * num_SMs) {
    return 1;
  }
  max_splits = std::min({max_splits, num_SMs, num_n_blocks});
  float max_efficiency = 0.f;
  std::vector<float> efficiency;
  efficiency.reserve(max_splits);
  auto ceildiv = [](int a, int b) { return (a + b - 1) / b; };
  // Some splits are not eligible. For example, if we have 64 blocks and choose
  // 11 splits, we'll have 6 * 10 + 4 blocks. If we choose 12 splits, we'll have
  // 6 * 11 + (-2) blocks (i.e. it's 11 splits anyway). So we check if the
  // number of blocks per split is the same as the previous num_splits.
  auto is_split_eligible = [&ceildiv, &num_n_blocks](int num_splits) {
    return num_splits == 1 ||
      ceildiv(num_n_blocks, num_splits) !=
      ceildiv(num_n_blocks, num_splits - 1);
  };
  for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
    if (!is_split_eligible(num_splits)) {
      efficiency.push_back(0.f);
    } else {
      float n_waves = float(batch_nheads_mblocks * num_splits) / num_SMs;
      float eff = n_waves / ceil(n_waves);
      // printf("num_splits = %d, eff = %f\n", num_splits, eff);
      if (eff > max_efficiency) {
        max_efficiency = eff;
      }
      efficiency.push_back(eff);
    }
  }
  for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
    if (!is_split_eligible(num_splits)) {
      continue;
    }
    if (efficiency[num_splits - 1] >= 0.85 * max_efficiency) {
      // printf("num_splits chosen = %d\n", num_splits);
      return num_splits;
    }
  }
  return 1;
}

void FlashAttnCuda(
  const NDArray& q, // batch_size x seqlen_q x num_heads x head_size
  const NDArray& k, // batch_size x seqlen_k x num_heads_k x head_size
  const NDArray& v, // batch_size x seqlen_k x num_heads_k x head_size
  NDArray& out_, // batch_size x seqlen_q x num_heads x head_size
  NDArray& q_padded, // batch_size x seqlen_q x num_heads x head_size_rounded
  NDArray& k_padded, // batch_size x seqlen_k x num_heads_k x head_size_rounded
  NDArray& v_padded, // batch_size x seqlen_k x num_heads_k x head_size_rounded
  NDArray& out_padded, // batch_size x seqlen_q x num_heads x head_size_rounded
  NDArray& softmax_lse, // batch_size × num_heads × seqlen_q
  NDArray& p, // batch_size × num_heads × seqlen_q_rounded × seqlen_k_rounded
  NDArray& rng_state, // 2  kCUDA  kInt64
  const float p_dropout, const float softmax_scale, const bool is_causal,
  const bool return_softmax, const Stream& stream) {
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  auto dprops = Device::dprop(q->device().index());
  // hipGetDeviceProperties(&dprops, q->device().index());
  // bool is_sm75 = dprops.major == 7 && dprops.minor == 5;
  bool is_sm8x = dprops.major == 8 && dprops.minor >= 0;
  bool is_sm90 = dprops.major == 9 && dprops.minor == 0;
  HT_ASSERT(is_sm90 || is_sm8x)
    << "FlashAttention only supports Ampere GPUs or newer.";
  // We will support Turing in the near future
  // HT_ASSERT(is_sm90 || is_sm8x || is_sm75, "FlashAttention only supports
  // Turing GPUs or newer.");

  auto q_dtype = q->dtype();
  HT_ASSERT(q_dtype == kFloat16 || q_dtype == kBFloat16)
    << "FlashAttention only support fp16 and bf16 data type";
  if (q_dtype == kBFloat16) {
    HT_ASSERT(is_sm90 || is_sm8x)
      << "bfloat16 is only supported on Ampere GPUs or newer";
  }
  HT_ASSERT(k->dtype() == q_dtype) << "query and key must have the same dtype";
  HT_ASSERT(v->dtype() == q_dtype)
    << "query and value must have the same dtype";

  HT_ASSERT(q->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(k->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(v->device().is_cuda()) << "Input tensor must be on CUDA device";

  HT_ASSERT(q->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(k->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(v->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";

  const auto sizes = q->shape();

  const int batch_size = sizes[0];
  const int seqlen_q = sizes[1];
  const int num_heads = sizes[2];
  const int head_size_og = sizes[3];
  const int seqlen_k = k->shape(1);
  const int num_heads_k = k->shape(2);
  HT_ASSERT(batch_size > 0) << "batch size must be postive";
  HT_ASSERT(head_size_og <= 256)
    << "FlashAttention forward only supports head dimension at most 256";
  HT_ASSERT(num_heads % num_heads_k == 0)
    << "Number of heads in key/value must divide number of heads in query";

  if (head_size_og % 8 != 0) {
    HTShape pad_shape = {0, 8 - head_size_og % 8};
    NDArray::pad(q, pad_shape, "constant", 0, stream.stream_index(), q_padded);
    NDArray::pad(k, pad_shape, "constant", 0, stream.stream_index(), k_padded);
    NDArray::pad(v, pad_shape, "constant", 0, stream.stream_index(), v_padded);
  } else {
    q_padded = q;
    k_padded = k;
    v_padded = v;
  }

  NDArray out;
  if (out_.is_defined()) {
    out = out_;
    HT_ASSERT(out->dtype() == q_dtype)
      << "Output must have the same dtype as inputs" << out->dtype() << " "
      << q_dtype;
    HT_ASSERT(out->device().is_cuda())
      << "Output tensor must be on CUDA device";
    HT_ASSERT(out->stride(-1) == 1)
      << "Output tensor must have contiguous last dimension";
    if (head_size_og % 8 != 0) {
      out = NDArray::empty_like(q_padded, stream.stream_index());
    }
  } else {
    out = NDArray::empty_like(q_padded, stream.stream_index());
  }

  auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
  const int head_size = round_multiple(head_size_og, 8);
  const int head_size_rounded = round_multiple(head_size, 32);
  const int seqlen_q_rounded = round_multiple(seqlen_q, 128);
  const int seqlen_k_rounded = round_multiple(seqlen_k, 128);

  // Otherwise the kernel will be launched from cuda:0 device
  // Cast to char to avoid compiler warning about narrowing

  // auto opts = q.options();

  // auto softmax_lse = NDArray::empty({batch_size, num_heads, seqlen_q},
  // q->device(), kFloat); NDArray p; Only return softmax if there's dropout to
  // reduce compilation time
  if (return_softmax) {
    HT_ASSERT(p_dropout > 0.0f)
      << "return_softmax is only supported when p_dropout > 0.0";
    p = NDArray::empty(
      {batch_size, num_heads, seqlen_q_rounded, seqlen_k_rounded}, q->device(),
      q->dtype(), stream.stream_index());
  }

  Flash_fwd_params params;
  set_params_fprop(
    params, batch_size, seqlen_q, seqlen_k, seqlen_q_rounded, seqlen_k_rounded,
    num_heads, num_heads_k, head_size, head_size_rounded, q_padded, k_padded,
    v_padded, out,
    /*cu_seqlens_q_d=*/nullptr,
    /*cu_seqlens_k_d=*/nullptr, return_softmax ? p->raw_data_ptr() : nullptr,
    softmax_lse->raw_data_ptr(), p_dropout, softmax_scale, is_causal);

  // This needs to match with run_mha_fwd_splitkv_dispatch
  const int block_n = is_sm90 || is_sm8x
    ? (head_size <= 64 ? 256 : (head_size <= 160 ? 128 : 64))
    : (head_size <= 64 ? 256 : (head_size <= 128 ? 128 : 64));
  const int num_n_blocks = (seqlen_k + block_n - 1) / block_n;
  // Technically kBlockM = 64 only for the splitKV kernels, not the standard
  // kernel. In any case we don't expect seqlen_q to be larger than 64 for
  // inference.
  const int num_m_blocks = (seqlen_q + 64 - 1) / 64;
  params.num_splits = 1;
  if (p_dropout == 0.0f) { // SplitKV is not implemented for dropout
    params.num_splits =
      num_splits_heuristic(batch_size * num_heads * num_m_blocks,
                           dprops.multiProcessorCount, num_n_blocks, 128);
    if (params.num_splits > 1) {
      NDArray softmax_lse_accum =
        NDArray::empty({params.num_splits, batch_size, num_heads, seqlen_q},
                       q->device(), kFloat, stream.stream_index());
      NDArray out_accum = NDArray::empty(
        {params.num_splits, batch_size, num_heads, seqlen_q, head_size_rounded},
        q->device(), kFloat, stream.stream_index());
      params.softmax_lseaccum_ptr = softmax_lse_accum->raw_data_ptr();
      params.oaccum_ptr = out_accum->raw_data_ptr();
    }
  }

  int64_t counter_offset = params.b * params.h * 32;
  params.rng_state = reinterpret_cast<uint64_t*>(rng_state->raw_data_ptr());

  if (p_dropout > 0.0) {
    params.philox_args = std::pair<uint64_t, uint64_t>(
      hydraulis::impl::GenNextRandomSeed(), counter_offset);
  }

  run_mha_fwd(params, hip_stream);

  if (head_size_og % 8 != 0) {
    HTShape pad_shape1 = {0, 8 - head_size_og % 8};
    NDArray::pad(out, pad_shape1, "constant", 0, stream.stream_index(),
                 out_padded);
  } else {
    out_padded = out;
    // NDArray::copy(out, stream.stream_index(), out_padded);
  }

  // if (head_size_og % 8 != 0) {
  //     out = out.index({"...", torch::indexing::Slice(torch::indexing::None,
  //     head_size_og)}); if (out_.is_defined()) { out_.value().copy_(out); }
  // }
  NDArray::MarkUsedBy({q, k, v, out_, q_padded, k_padded, v_padded, out_padded,
                       softmax_lse, p, rng_state},
                      stream);
}

void FlashAttnVarlenCuda(
  const NDArray& q, // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
  const NDArray& k, // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
  const NDArray& v, // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
  const NDArray& cu_seqlens_q, // b+1
  const NDArray& cu_seqlens_k, // b+1
  NDArray& out_, // total_q x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
  NDArray& q_padded, // batch_size x seqlen_q x num_heads x head_size_rounded
  NDArray& k_padded, // batch_size x seqlen_k x num_heads_k x head_size_rounded
  NDArray& v_padded, // batch_size x seqlen_k x num_heads_k x head_size_rounded
  NDArray& out_padded, // batch_size x seqlen_q x num_heads x head_size_rounded
  NDArray& softmax_lse, // batch_size × num_heads × seqlen_q
  NDArray& p, // batch_size × num_heads × seqlen_q_rounded × seqlen_k_rounded
  NDArray& rng_state, // 2  kCUDA  kInt64
  const int max_seqlen_q, const int max_seqlen_k, const float p_dropout,
  const float softmax_scale, const bool zero_tensors, const bool is_causal,
  const bool return_softmax, const Stream& stream) {
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  auto dprops = Device::dprop(q->device().index());
  // bool is_sm75 = dprops.major == 7 && dprops.minor == 5;
  bool is_sm8x = dprops.major == 8 && dprops.minor >= 0;
  bool is_sm90 = dprops.major == 9 && dprops.minor == 0;
  HT_ASSERT(is_sm90 || is_sm8x)
    << "FlashAttention only supports Ampere GPUs or newer.";
  // We will support Turing in the near future
  // HT_ASSERT(is_sm90 || is_sm8x || is_sm75, "FlashAttention only supports
  // Turing GPUs or newer.");

  auto q_dtype = q->dtype();
  HT_ASSERT(q_dtype == kFloat16 || q_dtype == kBFloat16)
    << "FlashAttention only support fp16 and bf16 data type";
  if (q_dtype == kBFloat16) {
    HT_ASSERT(is_sm90 || is_sm8x)
      << "bfloat16 is only supported on Ampere GPUs or newer";
  }
  HT_ASSERT(k->dtype() == q_dtype) << "query and key must have the same dtype";
  HT_ASSERT(v->dtype() == q_dtype)
    << "query and value must have the same dtype";
  HT_ASSERT(cu_seqlens_q->dtype() == kInt32)
    << "cu_seqlens_q must have dtype int32";
  HT_ASSERT(cu_seqlens_k->dtype() == kInt32)
    << "cu_seqlens_k must have dtype int32";

  HT_ASSERT(q->is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(k->is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(v->is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(cu_seqlens_q->is_cuda()) << "cu_seqlens_q must be on CUDA device";
  HT_ASSERT(cu_seqlens_k->is_cuda()) << "cu_seqlens_k must be on CUDA device";

  HT_ASSERT(q->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(k->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(v->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(cu_seqlens_q->is_contiguous()) << "cu_seqlens_q must be contiguous";
  HT_ASSERT(cu_seqlens_k->is_contiguous()) << "cu_seqlens_k must be contiguous";

  const auto sizes = q->shape();

  const int total_q = sizes[0];
  const int batch_size = cu_seqlens_q->numel() - 1;
  const int num_heads = sizes[1];
  const int head_size_og = sizes[2];
  const int total_k = k->shape(0);
  const int num_heads_k = k->shape(1);
  HT_ASSERT(batch_size > 0) << "batch size must be positive";
  HT_ASSERT(head_size_og <= 256)
    << "FlashAttention forward only supports head dimension at most 256";
  HT_ASSERT(num_heads % num_heads_k == 0)
    << "Number of heads in key/value must divide number of heads in query";

  if (head_size_og % 8 != 0) {
    HTShape pad_shape = {0, 8 - head_size_og % 8};
    NDArray::pad(q, pad_shape, "constant", 0, stream.stream_index(), q_padded);
    NDArray::pad(k, pad_shape, "constant", 0, stream.stream_index(), k_padded);
    NDArray::pad(v, pad_shape, "constant", 0, stream.stream_index(), v_padded);
  } else {
    q_padded = q;
    k_padded = k;
    v_padded = v;
  }

  NDArray out;
  if (out_.is_defined()) {
    out = out_;
    HT_ASSERT(out->dtype() == q_dtype)
      << "Output must have the same dtype as inputs" << out->dtype() << " "
      << q_dtype;
    HT_ASSERT(out->device().is_cuda())
      << "Output tensor must be on CUDA device";
    HT_ASSERT(out->stride(-1) == 1)
      << "Output tensor must have contiguous last dimension";
    if (head_size_og % 8 != 0) {
      out = NDArray::empty_like(q_padded, stream.stream_index());
    }
  } else {
    out = NDArray::empty_like(q_padded, stream.stream_index());
  }

  auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
  const int head_size = round_multiple(head_size_og, 8);
  const int head_size_rounded = round_multiple(head_size, 32);
  const int seqlen_q_rounded = round_multiple(max_seqlen_q, 128);
  const int seqlen_k_rounded = round_multiple(max_seqlen_k, 128);

  // auto opts = q.options();

  // auto softmax_lse =
  //   torch::empty({batch_size, num_heads, max_seqlen_q},
  //   opts.dtype(at::kFloat));
  // at::Tensor p;
  // Only return softmax if there's dropout to reduce compilation time
  if (return_softmax) {
    HT_ASSERT(p_dropout > 0.0f)
      << "return_softmax is only supported when p_dropout > 0.0";
    p = NDArray::empty(
      {batch_size, num_heads, seqlen_q_rounded, seqlen_k_rounded}, q->device(),
      q->dtype(), stream.stream_index());
  }

  if (zero_tensors) {
    NDArray::zeros_(out, stream.stream_index());
    NDArray::full_(softmax_lse, -std::numeric_limits<float>::infinity(),
                   stream.stream_index());
    if (return_softmax) {
      NDArray::zeros_(p, stream.stream_index());
    }
  }

  Flash_fwd_params params;
  set_params_fprop(
    params, batch_size, max_seqlen_q, max_seqlen_k, seqlen_q_rounded,
    seqlen_k_rounded, num_heads, num_heads_k, head_size, head_size_rounded,
    q_padded, k_padded, v_padded, out, cu_seqlens_q->raw_data_ptr(),
    cu_seqlens_k->raw_data_ptr(), return_softmax ? p->raw_data_ptr() : nullptr,
    softmax_lse->raw_data_ptr(), p_dropout, softmax_scale, is_causal);

  // number of times random will be generated per thread, to offset philox
  // counter in thc random state We use a custom RNG that increases the offset
  // by batch_size * nheads * 32.
  int64_t counter_offset = params.b * params.h * 32;
  params.rng_state = reinterpret_cast<uint64_t*>(rng_state->raw_data_ptr());
  // auto options =
  //   torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
  // auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
  // // Forward kernel will populate memory with the seed and offset.
  // params.rng_state = reinterpret_cast<uint64_t*>(rng_state.data_ptr());

  if (p_dropout > 0.0) {
    params.philox_args = std::pair<uint64_t, uint64_t>(
      hydraulis::impl::GenNextRandomSeed(), counter_offset);
  }

  run_mha_fwd(params, hip_stream);

  if (head_size_og % 8 != 0) {
    HTShape pad_shape1 = {0, 8 - head_size_og % 8};
    NDArray::pad(out, pad_shape1, "constant", 0, stream.stream_index(),
                 out_padded);
  } else {
    out_padded = out;
    // NDArray::copy(out, stream.stream_index(), out_padded);
  }

  // if (head_size_og % 8 != 0) {
  //     out = out.index({"...", torch::indexing::Slice(torch::indexing::None,
  //     head_size_og)}); if (out_.is_defined()) { out_.value().copy_(out); }
  // }
  NDArray::MarkUsedBy({q, k, v, cu_seqlens_q, cu_seqlens_k, out_, q_padded,
                       k_padded, v_padded, out_padded, softmax_lse, p,
                       rng_state},
                      stream);
}

void run_mha_bwd(Flash_bwd_params& params, hipStream_t stream,
                 const bool configure) {
  FP16_SWITCH(!params.is_bf16, [&] {
    if (params.d <= 32) {
      run_mha_bwd_<elem_type, 32>(params, stream, configure);
    } else if (params.d <= 64) {
      run_mha_bwd_<elem_type, 64>(params, stream, configure);
    } else if (params.d <= 96) {
      run_mha_bwd_<elem_type, 96>(params, stream, configure);
    } else if (params.d <= 128) {
      run_mha_bwd_<elem_type, 128>(params, stream, configure);
    } else if (params.d <= 160) {
      run_mha_bwd_<elem_type, 160>(params, stream, configure);
    } else if (params.d <= 192) {
      run_mha_bwd_<elem_type, 192>(params, stream, configure);
    } else if (params.d <= 224) {
      run_mha_bwd_<elem_type, 224>(params, stream, configure);
    } else if (params.d <= 256) {
      run_mha_bwd_<elem_type, 256>(params, stream, configure);
    }
  });
}

void FlashAttnGradientCuda(
  const NDArray& dout, // batch_size x seqlen_q x num_heads, x head_size_og
  const NDArray& q, // batch_size x seqlen_q x num_heads x head_size
  const NDArray& k, // batch_size x seqlen_k x num_heads_k x head_size
  const NDArray& v, // batch_size x seqlen_k x num_heads_k x head_size
  NDArray& out, // batch_size x seqlen_q x num_heads x head_size
  NDArray& softmax_lse, // b x h x seqlen_q
  NDArray& rng_state,
  NDArray& dq_, // batch_size x seqlen_q x num_heads x head_size
  NDArray& dk_, // batch_size x seqlen_k x num_heads_k x head_size
  NDArray& dv_, // batch_size x seqlen_k x num_heads_k x head_size
  const float p_dropout, // probability to drop
  const float softmax_scale, const bool is_causal, const Stream& stream) {
  auto dprops = Device::dprop(q->device().index());
  // bool is_sm75 = dprops.major == 7 && dprops.minor == 5;
  bool is_sm8x = dprops.major == 8 && dprops.minor >= 0;
  bool is_sm80 = dprops.major == 8 && dprops.minor == 0;
  bool is_sm90 = dprops.major == 9 && dprops.minor == 0;
  HT_ASSERT(is_sm90 || is_sm8x)
    << "FlashAttention only supports Ampere GPUs or newer.";
  // We will support Turing in the near future
  // HT_ASSERT(is_sm90 || is_sm8x || is_sm75, "FlashAttention only supports
  // Turing GPUs or newer.");

  bool is_dropout = p_dropout > 0.0;
  auto q_dtype = q->dtype();
  HT_ASSERT(q_dtype == kFloat16 || q_dtype == kBFloat16)
    << "FlashAttention only support fp16 and bf16 data type";
  if (q_dtype == kBFloat16) {
    HT_ASSERT(is_sm90 || is_sm8x)
      << "bfloat16 is only supported on Ampere GPUs or newer";
  }
  HT_ASSERT(k->dtype() == q_dtype) << "query and key must have the same dtype";
  HT_ASSERT(v->dtype() == q_dtype)
    << "query and value must have the same dtype";
  HT_ASSERT(out->dtype() == q_dtype)
    << "query and out must have the same dtype";
  HT_ASSERT(dout->dtype() == q_dtype)
    << "query and dout must have the same dtype";

  HT_ASSERT(q->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(k->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(v->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(out->device().is_cuda()) << "out tensor must be on CUDA device";
  HT_ASSERT(dout->device().is_cuda()) << "dout tensor must be on CUDA device";
  HT_ASSERT(softmax_lse->device().is_cuda())
    << "softmax_lse tensor must be on CUDA device";

  HT_ASSERT(q->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(k->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(v->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(out->stride(-1) == 1)
    << "out tensor must have contiguous last dimension";
  HT_ASSERT(dout->stride(-1) == 1)
    << "dout tensor must have contiguous last dimension";

  const auto sizes = q->shape();

  const int batch_size = sizes[0];
  const int seqlen_q = sizes[1];
  const int num_heads = sizes[2];
  const int head_size_og = dout->shape(3);
  const int head_size = sizes[3];
  const int seqlen_k = k->shape(1);
  const int num_heads_k = k->shape(2);
  HT_ASSERT(batch_size > 0) << "batch size must be positive";
  HT_ASSERT(head_size % 8 == 0) << "head_size should be a multiple of 8";
  HT_ASSERT(head_size <= 256)
    << "FlashAttention backward only supports head dimension at most 256";
  if (head_size > 192) {
    HT_ASSERT(is_sm80 || is_sm90)
      << "FlashAttention backward for head dim > 192 requires A100/A800 or H100/H800";
  }
  HT_ASSERT(num_heads % num_heads_k == 0)
    << "Number of heads in key/value must divide number of heads in query";

  auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
  const int head_size_rounded = round_multiple(head_size, 32);
  const int seqlen_q_rounded = round_multiple(seqlen_q, 128);
  const int seqlen_k_rounded = round_multiple(seqlen_k, 128);

  HT_ASSERT(head_size == round_multiple(head_size_og, 8))
    << "head_size must be head_size_og rounded to a multiple of 8";

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());

  NDArray dq, dk, dv;
  if (dq_.is_defined()) {
    dq = dq_;
    HT_ASSERT(dq->dtype() == q_dtype) << "dq must have the same dtype as q";
    HT_ASSERT(dq->device().is_cuda()) << "dq must be on CUDA device";
    HT_ASSERT(dq->stride(-1) == 1) << "dq must have contiguous last dimension";
    // CHECK_SHAPE(dq, batch_size, seqlen_q, num_heads, head_size);
  } else {
    dq = NDArray::empty_like(q);
  }
  if (dk_.is_defined()) {
    dk = dk_;
    HT_ASSERT(dk->dtype() == q_dtype) << "dk must have the same dtype as q";
    HT_ASSERT(dk->device().is_cuda()) << "dk must be on CUDA device";
    HT_ASSERT(dk->stride(-1) == 1) << "dk must have contiguous last dimension";
    // CHECK_SHAPE(dk, batch_size, seqlen_k, num_heads_k, head_size);
  } else {
    dk = NDArray::empty_like(k, stream.stream_index());
  }
  if (dv_.is_defined()) {
    dv = dv_;
    HT_ASSERT(dv->dtype() == q_dtype) << "dv must have the same dtype as q";
    HT_ASSERT(dv->device().is_cuda()) << "dv must be on CUDA device";
    HT_ASSERT(dv->stride(-1) == 1) << "dv must have contiguous last dimension";
    // CHECK_SHAPE(dv, batch_size, seqlen_k, num_heads_k, head_size);
  } else {
    dv = NDArray::empty_like(k, stream.stream_index());
  }

  NDArray dout_padded;
  if (head_size_og % 8 != 0) {
    HTShape pad_shape = {0, 8 - head_size_og % 8};
    NDArray::pad(dout, pad_shape, "constant", 0, stream.stream_index(),
                 dout_padded);
  } else {
    dout_padded = dout;
  }

  // bool loop = seqlen_k > blocksize_c;
  // TODO: change later, for now set to true for simplicity
  bool loop = true;

  // auto opts = q.options();
  auto softmax_d = NDArray::empty({batch_size, num_heads, seqlen_q_rounded},
                                  q->device(), kFloat, stream.stream_index());
  NDArray dq_accum;
  NDArray dk_accum, dv_accum;
  if (loop) {
    dq_accum = NDArray::empty(
      {batch_size, num_heads, seqlen_q_rounded, head_size_rounded}, q->device(),
      kFloat, stream.stream_index());
    // dk_accum = NDArray::empty({batch_size, num_heads_k, seqlen_k_rounded,
    // head_size_rounded}, opts->dtype(at::kFloat)); dv_accum =
    // NDArray::empty({batch_size, num_heads_k, seqlen_k_rounded,
    // head_size_rounded}, opts->dtype(at::kFloat));
  }

  NDArray dk_expanded, dv_expanded;
  if (num_heads_k != num_heads) { // MQA / GQA
    dk_expanded =
      NDArray::empty({batch_size, seqlen_k, num_heads, head_size}, q->device(),
                     q->dtype(), stream.stream_index());
    dv_expanded =
      NDArray::empty({batch_size, seqlen_k, num_heads, head_size}, q->device(),
                     q->dtype(), stream.stream_index());
  } else {
    dk_expanded = dk;
    dv_expanded = dv;
  }

  Flash_bwd_params params;

  set_params_dgrad(
    params, batch_size, seqlen_q, seqlen_k, seqlen_q_rounded, seqlen_k_rounded,
    num_heads, num_heads_k, head_size, head_size_rounded, q, k, v, out,
    dout_padded, dq, dk_expanded, dv_expanded, nullptr, nullptr,
    loop ? dq_accum->raw_data_ptr() : nullptr,
    // loop ? dk_accum->raw_data_ptr() : nullptr,
    // loop ? dv_accum->raw_data_ptr() : nullptr,
    nullptr, nullptr, softmax_lse->raw_data_ptr(), softmax_d->raw_data_ptr(),
    p_dropout, softmax_scale, is_causal);

  auto launch = &run_mha_bwd;
  // launch(params, stream, /*configure=*/true);

  // auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
  //     gen_, at::cuda::detail::getDefaultCUDAGenerator());

  // We use a custom RNG that increases the offset by batch_size * nheads * 32.
  int64_t counter_offset = params.b * params.h * 32;

  if (rng_state.is_defined()) {
    params.rng_state = reinterpret_cast<uint64_t*>(rng_state->raw_data_ptr());
  } else if (is_dropout) {
    // See Note [Acquire lock when using random generators]
    // std::lock_guard<std::mutex> lock(gen->mutex_);
    params.philox_args = std::pair<uint64_t, uint64_t>(
      hydraulis::impl::GenNextRandomSeed(), counter_offset);
    params.rng_state[0] = params.philox_args.first;
    params.rng_state[1] = params.philox_args.second;
  }

  launch(params, hip_stream, /*configure=*/false);

  // For MQA/GQA we need to sum dK and dV across the groups
  if (num_heads_k != num_heads) {
    NDArray::sum(NDArray::reshape(dk_expanded,
                                  {batch_size, seqlen_k, num_heads_k,
                                   num_heads / num_heads_k, head_size},
                                  stream.stream_index()),
                 {3}, false, stream.stream_index(), dk);
    NDArray::sum(NDArray::reshape(dv_expanded,
                                  {batch_size, seqlen_k, num_heads_k,
                                   num_heads / num_heads_k, head_size},
                                  stream.stream_index()),
                 {3}, false, stream.stream_index(), dv);
  }
  // if (head_size_og % 8 != 0) {
  //     dq = dq.index({"...", torch::indexing::Slice(torch::indexing::None,
  //     head_size_og)}); dk = dk.index({"...",
  //     torch::indexing::Slice(torch::indexing::None, head_size_og)}); dv =
  //     dv.index({"...", torch::indexing::Slice(torch::indexing::None,
  //     head_size_og)});
  // }

  NDArray::MarkUsedBy(
    {dout, q, k, v, out, softmax_lse, rng_state, dq_, dk_, dv_}, stream);
}

void FlashAttnVarlenGradientCuda(
  const NDArray& dout, // total_q x num_heads x head_size
  const NDArray& q, // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
  const NDArray& k, // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
  const NDArray& v, // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
  const NDArray& cu_seqlens_q, // b+1
  const NDArray& cu_seqlens_k, // b+1
  NDArray& out, // total_q x num_heads x head_size
  NDArray& softmax_lse, // b x h x s   softmax logsumexp
  NDArray& rng_state,
  NDArray& dq_, // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
  NDArray& dk_, // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
  NDArray& dv_, // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i
  const int max_seqlen_q,
  const int max_seqlen_k, // max sequence length to choose the kernel
  const float p_dropout, // probability to drop
  const float softmax_scale, const bool zero_tensors, 
	const bool is_causal, const Stream& stream) {
   auto dprops = Device::dprop(q->device().index());
  // bool is_sm75 = dprops.major == 7 && dprops.minor == 5;
  bool is_sm8x = dprops.major == 8 && dprops.minor >= 0;
  bool is_sm80 = dprops.major == 8 && dprops.minor == 0;
  bool is_sm90 = dprops.major == 9 && dprops.minor == 0;
  HT_ASSERT(is_sm90 || is_sm8x)
    << "FlashAttention only supports Ampere GPUs or newer.";
  // We will support Turing in the near future
  // HT_ASSERT(is_sm90 || is_sm8x || is_sm75, "FlashAttention only supports
  // Turing GPUs or newer.");

  bool is_dropout = p_dropout > 0.0;

auto q_dtype = q->dtype();
  HT_ASSERT(q_dtype == kFloat16 || q_dtype == kBFloat16)
    << "FlashAttention only support fp16 and bf16 data type";
  if (q_dtype == kBFloat16) {
    HT_ASSERT(is_sm90 || is_sm8x)
      << "bfloat16 is only supported on Ampere GPUs or newer";
  }
  HT_ASSERT(k->dtype() == q_dtype) << "query and key must have the same dtype";
  HT_ASSERT(v->dtype() == q_dtype)
    << "query and value must have the same dtype";
  HT_ASSERT(out->dtype() == q_dtype)
    << "query and out must have the same dtype";
  HT_ASSERT(dout->dtype() == q_dtype)
    << "query and dout must have the same dtype";

  HT_ASSERT(q->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(k->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(v->device().is_cuda()) << "Input tensor must be on CUDA device";
  HT_ASSERT(out->device().is_cuda()) << "out tensor must be on CUDA device";
  HT_ASSERT(dout->device().is_cuda()) << "dout tensor must be on CUDA device";
  HT_ASSERT(softmax_lse->device().is_cuda())
    << "softmax_lse tensor must be on CUDA device";
  HT_ASSERT(cu_seqlens_q->is_cuda())
	<< "cu_seqlens_q must be on CUDA device";
  HT_ASSERT(cu_seqlens_k->is_cuda())
	<< "cu_seqlens_k must be on CUDA device";

  HT_ASSERT(q->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(k->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(v->stride(-1) == 1)
    << "Input tensor must have contiguous last dimension";
  HT_ASSERT(out->stride(-1) == 1)
    << "out tensor must have contiguous last dimension";
  HT_ASSERT(dout->stride(-1) == 1)
    << "dout tensor must have contiguous last dimension";
  HT_ASSERT(cu_seqlens_q->is_contiguous())
		<< "cu_seqlens_q must be contiguous";
  HT_ASSERT(cu_seqlens_k->is_contiguous())
	  << "cu_seqlens_k must be contiguous";

  const auto sizes = q->shape();

  const int total_q = sizes[0];
  const int batch_size = cu_seqlens_q->numel() - 1;
  const int num_heads = sizes[1];
  const int head_size_og = dout->shape(2);
  const int head_size = sizes[2];
  const int total_k = k->shape(0);
  const int num_heads_k = k->shape(1);
  HT_ASSERT(batch_size > 0)
		<< "batch size must be positive";
  HT_ASSERT(head_size % 8 == 0)
		<< "head_size should be a multiple of 8";
  HT_ASSERT(head_size <= 256)
		<< "FlashAttention backward only supports head dimension at most 256";
  if (head_size > 192) {
    HT_ASSERT(is_sm80 || is_sm90)
			<< "FlashAttention backward for head dim > 192 requires A100/A800 or H100/H800";
  }
  HT_ASSERT(num_heads % num_heads_k == 0)
    << "Number of heads in key/value must divide number of heads in query";

  auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
  const int head_size_rounded = round_multiple(head_size, 32);
  const int seqlen_q_rounded = round_multiple(max_seqlen_q, 128);
  const int seqlen_k_rounded = round_multiple(max_seqlen_k, 128);

  HT_ASSERT(head_size == round_multiple(head_size_og, 8))
    << "head_size must be head_size_og rounded to a multiple of 8"
    << ", but found head_size = " << head_size << " and head_size_og = " << head_size_og;

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());

	NDArray dq, dk, dv;
  if (dq_.is_defined()) {
    dq = dq_;
    HT_ASSERT(dq->dtype() == q_dtype) << "dq must have the same dtype as q";
    HT_ASSERT(dq->device().is_cuda()) << "dq must be on CUDA device";
    HT_ASSERT(dq->stride(-1) == 1) << "dq must have contiguous last dimension";
  } else {
    dq = NDArray::empty_like(q);
  }
  if (dk_.is_defined()) {
    dk = dk_;
    HT_ASSERT(dk->dtype() == q_dtype) << "dk must have the same dtype as q";
    HT_ASSERT(dk->device().is_cuda()) << "dk must be on CUDA device";
    HT_ASSERT(dk->stride(-1) == 1) << "dk must have contiguous last dimension";
  } else {
    dk = NDArray::empty_like(k, stream.stream_index());
  }
  if (dv_.is_defined()) {
    dv = dv_;
    HT_ASSERT(dv->dtype() == q_dtype) << "dv must have the same dtype as q";
    HT_ASSERT(dv->device().is_cuda()) << "dv must be on CUDA device";
    HT_ASSERT(dv->stride(-1) == 1) << "dv must have contiguous last dimension";
  } else {
    dv = NDArray::empty_like(k, stream.stream_index());
  }

  NDArray dout_padded;
  if (head_size_og % 8 != 0) {
    HTShape pad_shape = {0, 8 - head_size_og % 8};
    NDArray::pad(dout, pad_shape, "constant", 0, stream.stream_index(),
                 dout_padded);
  } else {
    dout_padded = dout;
  }

  // bool loop = max_seqlen_k > blocksize_c;
  // TODO: change later, for now set to true for simplicity
  bool loop = true;

	auto softmax_d = NDArray::empty({batch_size, num_heads, seqlen_q_rounded},
                                  q->device(), kFloat, stream.stream_index());
  NDArray dq_accum;
  if (loop) {
    dq_accum = NDArray::empty(
      {batch_size, num_heads, seqlen_q_rounded, head_size_rounded}, q->device(),
      kFloat, stream.stream_index());
  }

  NDArray dk_expanded, dv_expanded;
  if (num_heads_k != num_heads) { // MQA / GQA
    dk_expanded =
      NDArray::empty({total_k, num_heads, head_size}, q->device(),
                     q->dtype(), stream.stream_index());
    dv_expanded =
      NDArray::empty({total_k, num_heads, head_size}, q->device(),
                     q->dtype(), stream.stream_index());
  } else {
    dk_expanded = dk;
    dv_expanded = dv;
  }

  if (zero_tensors) {
    NDArray::zeros_(dq, stream.stream_index());
		NDArray::zeros_(dk_expanded, stream.stream_index());
		NDArray::zeros_(dv_expanded, stream.stream_index());  
		NDArray::zeros_(softmax_d, stream.stream_index());
  }

  Flash_bwd_params params;

  set_params_dgrad(params, batch_size, max_seqlen_q, max_seqlen_k,
                   seqlen_q_rounded, seqlen_k_rounded, num_heads, num_heads_k,
                   head_size, head_size_rounded, q, k, v, out, dout_padded, dq,
                   dk_expanded, dv_expanded, cu_seqlens_q->raw_data_ptr(),
                   cu_seqlens_k->raw_data_ptr(),
                   loop ? dq_accum->raw_data_ptr() : nullptr, nullptr, nullptr,
                   softmax_lse->raw_data_ptr(), softmax_d->raw_data_ptr(), p_dropout,
                   softmax_scale, is_causal);

  auto launch = &run_mha_bwd;

  // We use a custom RNG that increases the offset by batch_size * nheads * 32.
  int64_t counter_offset = params.b * params.h * 32;

  if (rng_state.is_defined()) {
    params.rng_state = reinterpret_cast<uint64_t*>(rng_state->raw_data_ptr());
  } else if (is_dropout) {
    // See Note [Acquire lock when using random generators]
    // std::lock_guard<std::mutex> lock(gen->mutex_);
    params.philox_args = std::pair<uint64_t, uint64_t>(
      hydraulis::impl::GenNextRandomSeed(), counter_offset);
    params.rng_state[0] = params.philox_args.first;
    params.rng_state[1] = params.philox_args.second;
  }

  launch(params, hip_stream, /*configure=*/false);

  // For MQA/GQA we need to sum dK and dV across the groups
  if (num_heads_k != num_heads) {
    NDArray::sum(NDArray::reshape(dk_expanded,
                                  {total_k, num_heads_k, num_heads / num_heads_k, head_size},
                                  stream.stream_index()),
                 {2}, false, stream.stream_index(), dk);
    NDArray::sum(NDArray::reshape(dv_expanded,
                                  {total_k, num_heads_k, num_heads / num_heads_k, head_size},
                                  stream.stream_index()),
                 {2}, false, stream.stream_index(), dv);
  }
  // if (head_size_og % 8 != 0) {
  //     dq = dq.index({"...", torch::indexing::Slice(torch::indexing::None,
  //     head_size_og)}); dk = dk.index({"...",
  //     torch::indexing::Slice(torch::indexing::None, head_size_og)}); dv =
  //     dv.index({"...", torch::indexing::Slice(torch::indexing::None,
  //     head_size_og)});
  // }

  NDArray::MarkUsedBy(
    {dout, q, k, v, cu_seqlens_q, cu_seqlens_k,
		 out, softmax_lse, rng_state, dq_, dk_, dv_}, stream);
}

} // namespace impl
} // namespace hydraulis
