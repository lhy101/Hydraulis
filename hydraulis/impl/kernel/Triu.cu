#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t>
__global__ void triutril_kernel(const spec_t* input, spec_t* output, bool lower,
                                int64_t H, int64_t W, int64_t diagonal, size_t size,
                                const OffsetCalculator* in_offset_calculator,
                                const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int row = (idx / W) % H;
  int col = idx % W;
  bool mask = lower ? (col - row > diagonal) : (col - row < diagonal);
  auto in_offset = in_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(idx);
  spec_t zero = 0;
  output[out_offset] = mask ? zero : input[in_offset];
}

void TriuTrilCuda(const NDArray& input, NDArray& output, bool lower,
                  int64_t diagonal, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_SHAPE(input, output);

  size_t size = output->numel();
  int64_t ndim = input->ndim();
  int64_t H = input->shape(ndim - 2);
  int64_t W = input->shape(ndim - 1);
  if (diagonal < 0)
    diagonal += ndim;
  if (size == 0)
    return; 
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "TriuTrilCuda", [&]() {
      triutril_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), 
        lower, H, W, diagonal, size, in_offset_calculator,
        out_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
