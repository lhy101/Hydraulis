#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t>
__global__ void broadcast_kernel(const spec_t* input, size_t input_size,
                                 size_t size, spec_t* output,
                                 const OffsetCalculator* in_offset_calculator,
                                 const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    auto in_offset = in_offset_calculator->get(idx % input_size);
    auto out_offset = out_offset_calculator->get(idx);
    output[out_offset] = input[in_offset];
  }
}

template <typename spec_t>
__global__ void broadcast_gradient_kernel(const spec_t* input, size_t input_size,
                                          size_t size, spec_t* output,
                                          const OffsetCalculator* in_offset_calculator,
                                          const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    auto in_offset = in_offset_calculator->get(idx);
    auto out_offset = out_offset_calculator->get(idx);
    output[out_offset] = input[in_offset];
  }
}

void BroadcastCuda(const NDArray& input, NDArray& output,
                   const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = output->numel();
  size_t input_size = input->numel();
  if (size == 0 || input_size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "BroadcastCuda", [&]() {
      broadcast_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), input_size, size,
        output->data_ptr<spec_t>(), in_offset_calculator,
        out_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

void BroadcastGradientCuda(const NDArray& input, NDArray& output,
                           const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = output->numel();
  size_t input_size = input->numel();
  if (size == 0 || input_size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "BroadcastGradientCuda", [&]() {
      broadcast_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), input_size, size,
        output->data_ptr<spec_t>(), in_offset_calculator,
        out_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
