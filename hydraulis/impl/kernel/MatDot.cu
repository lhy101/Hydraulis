#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t>
__global__ void dot_kernel(const spec_t* inputA, const spec_t* inputB,
                           size_t size, size_t size2, spec_t* output,
                           const OffsetCalculator* A_offset_calculator,
                           const OffsetCalculator* B_offset_calculator,
                           const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    auto A_offset = A_offset_calculator->get(idx);
    auto B_offset = B_offset_calculator->get((int) (idx % size2));
    auto out_offset = out_offset_calculator->get(idx);
    output[out_offset] = inputA[A_offset] * inputB[B_offset];
  }
}

void MatDotCuda(const NDArray& inputA, const NDArray& inputB, NDArray& output,
                const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(inputA);
  HT_ASSERT_SAME_DEVICE(inputA, output);
  HT_ASSERT_SAME_DEVICE(inputB, output);
  HT_ASSERT_SAME_SHAPE(inputA, output);

  size_t size = inputA->numel();
  size_t size2 = inputB->numel();
  if (size == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray A_offset_calculator_arr, B_offset_calculator_arr,
          out_offset_calculator_arr;
  OffsetCalculator *A_offset_calculator, *B_offset_calculator,
                   *out_offset_calculator;
  std::tie(A_offset_calculator_arr, A_offset_calculator) =
    AllocOffsetCalculator(inputA, stream);
  std::tie(B_offset_calculator_arr, B_offset_calculator) = 
    AllocOffsetCalculator(inputB, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    inputA->dtype(), spec_t, "MatDotCuda", [&]() {
      dot_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        inputA->data_ptr<spec_t>(), inputB->data_ptr<spec_t>(), size, size2,
        output->data_ptr<spec_t>(), A_offset_calculator,
        B_offset_calculator, out_offset_calculator);
    });
  NDArray::MarkUsedBy({inputA, inputB, output, A_offset_calculator_arr,
                      B_offset_calculator_arr, out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
