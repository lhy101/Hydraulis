#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t>
__global__ void dynamic_concatenate_kernel(const spec_t* input, spec_t* output,
                                   int dynamic_input_width, int input_width, int output_width,
                                   int offset, int concat_size, size_t size,
                                   const OffsetCalculator* in_offset_calculator,
                                   const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  int post_ind = idx % concat_size;
  int prev_ind = idx / concat_size;
  if (prev_ind % input_width >= dynamic_input_width) // the paddings
    return;
  int mid_ind = prev_ind % input_width + offset;
  prev_ind = prev_ind / input_width;
  int out_ind = (prev_ind * output_width + mid_ind) * concat_size + post_ind;
  auto in_offset = in_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(out_ind);
  output[out_offset] = input[in_offset];
}

void DynamicConcatenateCuda(const NDArray& input, NDArray& output, size_t axis,
                     size_t offset, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);

  size_t size = input->numel();
  int now_ndim = output->ndim();
  HT_ASSERT(input->ndim() == now_ndim);
  int num_concats = 1;
  for (int i = 0; i < axis; ++i) {
    int cur_dim = output->shape(i);
    HT_ASSERT(input->dynamic_shape(i) == cur_dim);
    num_concats *= cur_dim;
  }
  int concat_size = 1;
  for (int i = axis + 1; i < now_ndim; ++i) {
    int cur_dim = output->shape(i);
    HT_ASSERT(input->dynamic_shape(i) == cur_dim);
    concat_size *= cur_dim;
  }
  int input_width = input->shape(axis);
  int dynamic_input_width = input->dynamic_shape(axis);
  int output_width = output->shape(axis);
  if (size == 0 || input_width == 0 || output_width == 0)
    return;
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "DynamicConcatenateCuda", [&]() {
      dynamic_concatenate_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), 
        dynamic_input_width, input_width, output_width, offset, concat_size, size,
        in_offset_calculator, out_offset_calculator);
    });
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
