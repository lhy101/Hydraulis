#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/kernel/Reduce.cuh"

namespace hydraulis {
namespace impl {

template <typename acc_t>
struct SumOp {
  __device__ __forceinline__ acc_t operator()(acc_t a, acc_t b) const {
    return a + b;
  }
};

template <typename spec_t, typename acc_t = spec_t, typename out_t = spec_t>
struct sum_functor {
  void operator()(const NDArray& in_arr, NDArray& out_arr, const int64_t* axes,
                  int64_t num_ax, const Stream& stream) {
    launch_reduce_kernel<spec_t, out_t, acc_t>(in_arr, out_arr, axes, num_ax,
                                               func_wrapper<out_t, acc_t>(SumOp<acc_t>()),
                                               0., stream);
  }
};

void ReduceSumCuda(const NDArray& in_arr, NDArray& out_arr, const int64_t* axes,
                   int64_t num_ax, const Stream& stream) {
  CUDAStream hip_stream(stream);
  if (out_arr->dtype() == DataType::FLOAT16) {
    sum_functor<hydraulis::float16, float>{}(in_arr, out_arr, axes, num_ax, stream);
  } else if (in_arr->dtype() == DataType::FLOAT16 && out_arr->dtype() == DataType::FLOAT32) {
    sum_functor<hydraulis::float16, float, float>{}(in_arr, out_arr, axes, num_ax, stream);
  } else if (out_arr->dtype() == DataType::BFLOAT16) {
    sum_functor<hydraulis::bfloat16, float>{}(in_arr, out_arr, axes, num_ax, stream);
  } else if (in_arr->dtype() == DataType::BFLOAT16 && out_arr->dtype() == DataType::FLOAT32) {
    sum_functor<hydraulis::bfloat16, float, float>{}(in_arr, out_arr, axes, num_ax, stream);
  } else {
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
      out_arr->dtype(), spec_t, "ReduceSumCuda", [&]() {
        sum_functor<spec_t>{}(in_arr, out_arr, axes, num_ax, stream);
      });
  }
}

} // namespace impl
} // namespace hydraulis