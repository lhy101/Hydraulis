#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t>
__global__ void
pad_kernel(const spec_t* input_data, spec_t* output_data, size_t begin_N,
           size_t end_N, size_t N, size_t begin_C, size_t end_C, size_t C,
           size_t begin_H, size_t end_H, size_t H, size_t begin_W,
           size_t end_W, size_t W, spec_t constant_value,
           const OffsetCalculator* in_offset_calculator,
           const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N * C * H * W)
    return;
  size_t idx_N = idx / (C * H * W);
  size_t idx_C = idx % (C * H * W) / (H * W);
  size_t idx_H = idx % (H * W) / W;
  size_t idx_W = idx % W;
  auto out_offset = out_offset_calculator->get(idx);
  if (idx_N >= begin_N && idx_N < end_N && idx_C >= begin_C && idx_C < end_C &&
      idx_H >= begin_H && idx_H < end_H && idx_W >= begin_W && idx_W < end_W) {
    auto in_offset = in_offset_calculator->get(
      (((idx_N - begin_N) * (end_C - begin_C) + idx_C - begin_C)
      * (end_H - begin_H) + idx_H - begin_H) * (end_W - begin_W) + idx_W - begin_W);
    output_data[out_offset] = input_data[in_offset];
  } else {
    output_data[out_offset] = constant_value;
  }
}

template <typename spec_t>
__global__ void
pad_gradient_kernel(const spec_t* output_grad, spec_t* input_grad, size_t N,
                    size_t C, size_t H, size_t W, size_t begin_N,
                    size_t begin_C, size_t begin_H, size_t begin_W,
                    size_t out_N, size_t out_C, size_t out_H, size_t out_W,
                    const OffsetCalculator* out_grad_offset_calculator,
                    const OffsetCalculator* in_grad_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N * C * H * W)
    return;
  size_t idx_N = idx / (C * H * W);
  size_t idx_C = idx % (C * H * W) / (H * W);
  size_t idx_H = idx % (H * W) / W;
  size_t idx_W = idx % W;
  auto in_grad_offset = in_grad_offset_calculator->get(idx);
  auto out_grad_offset = out_grad_offset_calculator->get(
    (((idx_N + begin_N) * out_C + idx_C + begin_C) * out_H + idx_H + begin_H) * out_W +
    idx_W + begin_W);
  input_grad[in_grad_offset] = output_grad[out_grad_offset];
}

void PadCuda(const NDArray& input, NDArray& output, const HTShape& paddings,
             const Stream& stream, std::string mode = "constant",
             double constant_values = 0) {
  HT_ASSERT(input->is_cuda()) << "Input is not on a host device.";
  HT_ASSERT(output->is_cuda()) << "Output is not on a host device.";
  HT_ASSERT(input->device() == output->device())
    << "Input and output are not on the same host device. "
    << "Devices: (input) " << input->device() << " vs. (output) "
    << output->device();
  size_t pad_len = paddings.size();
  size_t len = pad_len;
  size_t endpoint[8];
  for (int i = 0; i < 4; i++) {
    if (i < (4 - len / 2)) {
      HT_ASSERT((input->shape(i)) == (output->shape(i)));
      endpoint[i * 2] = 0;
      endpoint[i * 2 + 1] = input->shape(i);
    } else {
      HT_ASSERT((input->shape(i) + paddings[(i - (4 - len / 2)) * 2] +
                 paddings[(i - (4 - len / 2)) * 2 + 1]) == (output->shape(i)))
	<< "input shape = " << input->shape() << ", ouput shape = " << output->shape() << ", paddings = " << paddings;
      endpoint[i * 2] = paddings[(i - (4 - len / 2)) * 2];
      endpoint[i * 2 + 1] = paddings[(i - (4 - len / 2)) * 2] + input->shape(i);
    }
  }
  size_t size = output->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  if (mode == "constant") {
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
      input->dtype(), spec_t, "PadCuda", [&]() {
        pad_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
          input->data_ptr<spec_t>(), output->data_ptr<spec_t>(), endpoint[0],
          endpoint[1], output->shape(0), endpoint[2], endpoint[3],
          output->shape(1), endpoint[4], endpoint[5], output->shape(2),
          endpoint[6], endpoint[7], output->shape(3), constant_values,
          in_offset_calculator, out_offset_calculator);
      });
  }
  NDArray::MarkUsedBy({input, output, in_offset_calculator_arr,
                      out_offset_calculator_arr}, stream);
}

void PadGradientCuda(const NDArray& output_grad, NDArray& input_grad,
                     const HTShape& paddings, const Stream& stream,
                     std::string mode = "constant") {
  HT_ASSERT(output_grad->is_cuda()) << "Output_grad is not on a host device.";
  HT_ASSERT(input_grad->is_cuda()) << "Input_grad is not on a host device.";
  HT_ASSERT(input_grad->device() == output_grad->device())
    << "input and output grads are not on the same host device. "
    << "Devices: (input_grad) " << input_grad->device() << " vs. (output_grad) "
    << output_grad->device();
  size_t pad_len = paddings.size();
  size_t len = pad_len;
  size_t begin_p[4];
  size_t N = input_grad->shape(0);
  size_t C = input_grad->shape(1);
  size_t H = input_grad->shape(2);
  size_t W = input_grad->shape(3);

  size_t out_N = output_grad->shape(0);
  size_t out_C = output_grad->shape(1);
  size_t out_H = output_grad->shape(2);
  size_t out_W = output_grad->shape(3);

  for (int i = 0; i < 4; i++) {
    if (i < (4 - len / 2)) {
      begin_p[i] = 0;
    } else {
      begin_p[i] = paddings[(i - (4 - len / 2)) * 2];
    }
  }
  size_t size = input_grad->numel();
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray out_grad_offset_calculator_arr, in_grad_offset_calculator_arr;
  OffsetCalculator *out_grad_offset_calculator, *in_grad_offset_calculator;
  std::tie(out_grad_offset_calculator_arr, out_grad_offset_calculator) =
    AllocOffsetCalculator(output_grad, stream);
  std::tie(in_grad_offset_calculator_arr, in_grad_offset_calculator) = 
    AllocOffsetCalculator(input_grad, stream);
  if (mode == "constant") {
    HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
      input_grad->dtype(), spec_t, "PadGradientCuda", [&]() {
        pad_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
          output_grad->data_ptr<spec_t>(), input_grad->data_ptr<spec_t>(), N, C,
          H, W, begin_p[0], begin_p[1], begin_p[2], begin_p[3], out_N, out_C,
          out_H, out_W, out_grad_offset_calculator, in_grad_offset_calculator);
      });
  }
  NDArray::MarkUsedBy({input_grad, output_grad, out_grad_offset_calculator_arr,
                      in_grad_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
