#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/cuda/CUDARand.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"

namespace hydraulis {
namespace impl {

template <typename spec_t, typename mask_t>
__global__ void dropout2d_kernel(const spec_t* input, spec_t* output, mask_t* mask,
                                 float drop_rate, size_t size,
                                 size_t last_two,
                                 const OffsetCalculator* in_offset_calculator,
                                 const OffsetCalculator* out_offset_calculator,
                                 const OffsetCalculator* mask_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t leader = size_t(idx / last_two) * last_two;
  auto out_offset = out_offset_calculator->get(leader);
  mask_t keep_mask = (mask_t) (output[out_offset] >= drop_rate);
  auto in_offset = in_offset_calculator->get(idx);
  out_offset = out_offset_calculator->get(idx);
  auto mask_offset = mask_offset_calculator->get(idx);
  output[out_offset] = input[in_offset] * keep_mask / (1 - drop_rate);
  mask[mask_offset] = keep_mask;
}

template <typename spec_t, typename mask_t>
__global__ void dropout2d_gradient_kernel(const spec_t* grad,
                                          const mask_t* fw_mask, spec_t* output,
                                          float drop_rate, size_t size,
                                          const OffsetCalculator* grad_offset_calculator,
                                          const OffsetCalculator* fw_mask_offset_calculator,
                                          const OffsetCalculator* out_offset_calculator) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  auto mask_offset = fw_mask_offset_calculator->get(idx);
  auto grad_offset = grad_offset_calculator->get(idx);
  auto out_offset = out_offset_calculator->get(idx);
  output[out_offset] = grad[grad_offset] * fw_mask[mask_offset]  / (1 - drop_rate);
}

void Dropout2dCuda(const NDArray& input, double drop_rate, uint64_t seed,
                   NDArray& output, NDArray& mask, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_DEVICE(input, mask);
  HT_ASSERT_SAME_SHAPE(input, output);
  HT_ASSERT_SAME_SHAPE(input, mask);
  size_t size = input->numel();
  if (size == 0)
    return;
  HT_ASSERT(input->ndim() == 4);
  size_t last_two_size = 1;
  last_two_size *= input->shape(input->ndim() - 1);
  last_two_size *= input->shape(input->ndim() - 2);

  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray in_offset_calculator_arr, out_offset_calculator_arr,
          mask_offset_calculator_arr;
  OffsetCalculator *in_offset_calculator, *out_offset_calculator,
                   *mask_offset_calculator;
  std::tie(in_offset_calculator_arr, in_offset_calculator) =
    AllocOffsetCalculator(input, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  std::tie(mask_offset_calculator_arr, mask_offset_calculator) =
    AllocOffsetCalculator(mask, stream);
  hiprandGenerator_t gen;
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  CURAND_CALL(hiprandSetStream(gen, hip_stream));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
  HT_DISPATCH_FLOATING_TYPES(input->dtype(), spec_t, "Dropout2dCuda", [&]() {
    curand_gen_uniform<spec_t>(gen, output->data_ptr<spec_t>(), size);
    dropout2d_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
      input->data_ptr<spec_t>(), output->data_ptr<spec_t>(),
      mask->data_ptr<bool>(), static_cast<float>(drop_rate), size,
      last_two_size, in_offset_calculator,
      out_offset_calculator, mask_offset_calculator);
  });
  CURAND_CALL(hiprandDestroyGenerator(gen));
  NDArray::MarkUsedBy({input, output, mask, in_offset_calculator_arr,
                      out_offset_calculator_arr, mask_offset_calculator_arr}, stream);
}

void Dropout2dGradientCuda(const NDArray& grad, const NDArray& fw_mask,
                           double drop_rate, NDArray& output,
                           const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(grad);
  HT_ASSERT_SAME_DEVICE(grad, output);
  HT_ASSERT_SAME_DEVICE(grad, fw_mask);
  HT_ASSERT_SAME_SHAPE(grad, output);
  HT_ASSERT_SAME_SHAPE(grad, fw_mask);
  size_t size = grad->numel();
  if (size == 0)
    return;

  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  NDArray grad_offset_calculator_arr, fw_mask_offset_calculator_arr,
          out_offset_calculator_arr;
  OffsetCalculator *grad_offset_calculator, *fw_mask_offset_calculator,
                   *out_offset_calculator;
  std::tie(grad_offset_calculator_arr, grad_offset_calculator) =
    AllocOffsetCalculator(grad, stream);
  std::tie(fw_mask_offset_calculator_arr, fw_mask_offset_calculator) =
    AllocOffsetCalculator(fw_mask, stream);
  std::tie(out_offset_calculator_arr, out_offset_calculator) = 
    AllocOffsetCalculator(output, stream);
  HT_DISPATCH_FLOATING_TYPES(grad->dtype(), spec_t, "Dropout2dGradientCuda", [&]() {
    dropout2d_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
      grad->data_ptr<spec_t>(), fw_mask->data_ptr<spec_t>(),
      output->data_ptr<spec_t>(), static_cast<float>(drop_rate), size,
      grad_offset_calculator, fw_mask_offset_calculator,
      out_offset_calculator);
  });
  NDArray::MarkUsedBy({grad, fw_mask, output, grad_offset_calculator_arr,
                      fw_mask_offset_calculator_arr, out_offset_calculator_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
