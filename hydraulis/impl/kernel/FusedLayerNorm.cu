#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/core/memory_pool.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/cuda/CUDADnn.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/kernel/Binary.cuh"
#include "hydraulis/impl/utils/cuda_math.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"
#include <chrono>

namespace hydraulis {
namespace impl {

template <typename T>
__device__ __forceinline__ T WARP_SHFL(T value, int srcLane, int width = warpSize, unsigned int mask = 0xffffffff)
{
#if !defined(USE_ROCM)
    return __shfl_sync(mask, value, srcLane, width);
#else
    return __shfl(value, srcLane, width);
#endif
}

template <typename T>
__device__ __forceinline__ T WARP_SHFL_XOR(T value, int laneMask, int width = warpSize, unsigned int mask = 0xffffffff)
{
#if !defined(USE_ROCM)
    return __shfl_xor_sync(mask, value, laneMask, width);
#else
    return __shfl_xor(value, laneMask, width);
#endif
}

template<typename U> __device__
void cuWelfordOnlineSum(
  const U curr,
  U& mu,
  U& sigma2,
  U& count)
{
  count = count + U(1);
  U delta = curr - mu;
  U lmean = mu + delta / count;
  mu = lmean;
  U delta2 = curr - lmean;
  sigma2 = sigma2 + delta * delta2;
}

template<typename U> __device__
void cuChanOnlineSum(
  const U muB,
  const U sigma2B,
  const U countB,
  U& mu,
  U& sigma2,
  U& count)
{
  U delta = muB - mu;
  U nA = count;
  U nB = countB;
  count = count + countB;
  U nX = count;
  if (nX > U(0)) {
    nA = nA / nX;
    nB = nB / nX;
    mu = nA*mu + nB*muB;
    sigma2 = sigma2 + sigma2B + delta * delta * nA * nB * nX;
  } else {
    mu = U(0);
    sigma2 = U(0);
  }
}

template<typename U> __device__
void cuRMSOnlineSum(
  const U curr,
  U& sigma2)
{
  sigma2 = sigma2 + curr * curr;
}

template<typename U> __device__
void cuChanRMSOnlineSum(
  const U sigma2B,
  U& sigma2)
{
  sigma2 = sigma2 + sigma2B;
}


template<typename T, typename U> __device__
void cuWelfordMuSigma2(
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const int i1,
  U& mu,
  U& sigma2,
  U* buf,
  bool rms_only)
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  U count = U(0);
  mu= U(0);
  sigma2 = U(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const T* lvals = vals + i1*n2;
    int l = 4*thrx;
    for (;  l+3 < n2;  l+=4*numx) {
      for (int k = 0;  k < 4;  ++k) {
        U curr = static_cast<U>(lvals[l+k]);
        if (!rms_only) {
          cuWelfordOnlineSum<U>(curr,mu,sigma2,count);
        } else {
          cuRMSOnlineSum<U>(curr, sigma2);
        }
      }
    }
    for (;  l < n2;  ++l) {
      U curr = static_cast<U>(lvals[l]);
      if (!rms_only) {
        cuWelfordOnlineSum<U>(curr,mu,sigma2,count);
      } else {
       cuRMSOnlineSum<U>(curr, sigma2);
      }
    }
    // intra-warp reductions
    for (int l = 0;  l <= 4;  ++l) {
      int srcLaneB = (threadIdx.x+(1<<l))&31;
      U sigma2B = WARP_SHFL(sigma2, srcLaneB);
      if (!rms_only) {
        U muB = WARP_SHFL(mu, srcLaneB);
        U countB = WARP_SHFL(count, srcLaneB);
        cuChanOnlineSum<U>(muB,sigma2B,countB,mu,sigma2,count);
      } else {
        cuChanRMSOnlineSum<U>(sigma2B, sigma2);
      }
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      U* ubuf = (U*)buf;
      U* ibuf = (U*)(ubuf + blockDim.y);
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_y = threadIdx.y - offset;
          if (!rms_only) {
            ubuf[2*wrt_y] = mu;
            ibuf[wrt_y] = count;
          }
          ubuf[2*wrt_y+1] = sigma2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          U sigma2B = ubuf[2*threadIdx.y+1];
          if (!rms_only) {
            U muB = ubuf[2*threadIdx.y];
            U countB = ibuf[threadIdx.y];
            cuChanOnlineSum<U>(muB,sigma2B,countB,mu,sigma2,count);
          } else {
            cuChanRMSOnlineSum<U>(sigma2B,sigma2);
          }
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        if (!rms_only) {
          ubuf[0] = mu;
        }
        ubuf[1] = sigma2;
      }
      __syncthreads();
      if (!rms_only) {
        mu = ubuf[0];
      }
      sigma2 = ubuf[1]/U(n2);
      // don't care about final value of count, we know count == n2
    } else {
      if (!rms_only) {
        mu = WARP_SHFL(mu, 0);
      }
      sigma2 = WARP_SHFL(sigma2/U(n2), 0);
    }
  }
}

template<> __device__
void cuWelfordMuSigma2(
  const float16* __restrict__ vals,
  const int n1,
  const int n2,
  const int i1,
  float& mu,
  float& sigma2,
  float* buf,
  bool rms_only)
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  float count = 0.0f;
  mu= float(0);
  sigma2 = float(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const float16* lvals = vals + i1*n2;
    int l = 8*thrx;
    if ((((size_t)lvals)&3) != 0) {
      // 16 bit alignment
      // first thread consumes first point
      if (thrx == 0) {
        float curr = static_cast<float>(lvals[0]);
        if (!rms_only) {
          cuWelfordOnlineSum(curr,mu,sigma2,count);
        } else {
          cuRMSOnlineSum(curr, sigma2);
        }

      }
      ++l;
    }
    // at this point, lvals[l] are 32 bit aligned for all threads.
    for (;  l+7 < n2;  l+=8*numx) {
      for (int k = 0;  k < 8;  k+=2) {
        float2 curr = __half22float2(*((__half2*)(lvals+l+k)));
        if (!rms_only) {
          cuWelfordOnlineSum(curr.x,mu,sigma2,count);
          cuWelfordOnlineSum(curr.y,mu,sigma2,count);
        } else {
          cuRMSOnlineSum(curr.x, sigma2);
          cuRMSOnlineSum(curr.y, sigma2);
        }
      }
    }
    for (;  l < n2;  ++l) {
      float curr = static_cast<float>(lvals[l]);
      if (!rms_only) {
        cuWelfordOnlineSum(curr,mu,sigma2,count);
      } else {
        cuRMSOnlineSum(curr, sigma2);
      }
    }
    // intra-warp reductions
    for (int l = 0;  l <= 4;  ++l) {
      int srcLaneB = (threadIdx.x+(1<<l))&31;
      float sigma2B = WARP_SHFL(sigma2, srcLaneB);
      if (!rms_only) {
        float muB = WARP_SHFL(mu, srcLaneB);
        float countB = WARP_SHFL(count, srcLaneB);
        cuChanOnlineSum(muB,sigma2B,countB,mu,sigma2,count);
      } else {
        cuChanRMSOnlineSum(sigma2B, sigma2);
      }
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      float* ubuf = (float*)buf;
      float* ibuf = (float*)(ubuf + blockDim.y);
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_y = threadIdx.y - offset;
          ubuf[2*wrt_y+1] = sigma2;
          if (!rms_only) {
            ubuf[2*wrt_y] = mu;
            ibuf[wrt_y] = count;
          }
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          float sigma2B = ubuf[2*threadIdx.y+1];
          if (!rms_only) {
            float muB = ubuf[2*threadIdx.y];
            float countB = ibuf[threadIdx.y];
            cuChanOnlineSum(muB,sigma2B,countB,mu,sigma2,count);
          } else {
            cuChanRMSOnlineSum(sigma2B, sigma2);
          }
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        if (!rms_only) {
          ubuf[0] = mu;
        }
        ubuf[1] = sigma2;
      }
      __syncthreads();
      if (!rms_only) {
        mu = ubuf[0];
      }
      sigma2 = ubuf[1]/float(n2);
      // don't care about final value of count, we know count == n2
    } else {
      if (!rms_only) {
        mu = WARP_SHFL(mu, 0);
      }
      sigma2 = WARP_SHFL(sigma2/float(n2), 0);
    }
  }
}

template<typename U> __device__ U rsqrt(U v) {
  return U(1) / hydraulis::cuda::cuda_sqrt(v);
}
template<> __device__ float rsqrt(float v) {
  return rsqrtf(v);
}
template<> __device__ double rsqrt(double v) {
  return rsqrt(v);
}

namespace {
// This is the un-specialized struct.  Note that we prevent instantiation of this
// struct by putting an undefined symbol in the function body so it won't compile.
//  template <typename T>
//  struct SharedMemory
//  {
//      // Ensure that we won't compile any un-specialized types
//      __device__ T *getPointer()
//      {
//          extern __device__ void error(void);
//          error();
//          return NULL;
//      }
//  };
// https://github.com/NVIDIA/apex/issues/246
template <typename T>
struct SharedMemory;

template <>
struct SharedMemory <float16>
{
    __device__ float16 *getPointer()
    {
        extern __shared__ __half s_float16[];
        return (float16*)s_float16;
    }
};

template <>
struct SharedMemory <bfloat16>
{
    __device__ bfloat16 *getPointer()
    {
        extern __shared__ __hip_bfloat16 s_bfloat16[];
        return (bfloat16*)s_bfloat16;
    }
};

template <>
struct SharedMemory <float>
{
    __device__ float *getPointer()
    {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template <>
struct SharedMemory <double>
{
    __device__ double *getPointer()
    {
        extern __shared__ double s_double[];
        return s_double;
    }
};
}

template<typename T, typename U, typename V> __device__
void cuApplyLayerNorm_(
  V* __restrict__ output_vals,
  U* __restrict__ mean,
  U* __restrict__ invvar,
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const U epsilon,
  const V* __restrict__ gamma,
  const V* __restrict__ beta,
  bool rms_only
  )
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensors are contiguous
  //
  for (auto i1=blockIdx.y; i1 < n1; i1 += gridDim.y) {
    SharedMemory<U> shared;
    U* buf = shared.getPointer();
    U mu,sigma2;
    cuWelfordMuSigma2(vals,n1,n2,i1,mu,sigma2,buf,rms_only);

    const T* lvals = vals + i1*n2;
    V* ovals = output_vals + i1*n2;
    U c_invvar = rsqrt(sigma2 + epsilon);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL && (beta != NULL || rms_only)) {
      for (int i = thrx;  i < n2;  i+=numx) {
        U curr = static_cast<U>(lvals[i]);
        if (!rms_only) {
          ovals[i] = gamma[i] * static_cast<V>(c_invvar * (curr - mu)) + beta[i];
        } else {
          ovals[i] = gamma[i] * static_cast<V>(c_invvar * curr);
        }

      }
    } 
    else {
      for (int i = thrx;  i < n2;  i+=numx) {
        U curr = static_cast<U>(lvals[i]);
        if (!rms_only) {
          ovals[i] = static_cast<V>(c_invvar * (curr - mu));
        } else {
          ovals[i] = static_cast<V>(c_invvar * curr);
        }
      }
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      if (!rms_only) {
        mean[i1] = mu;
      }
      invvar[i1] = c_invvar;
    }
    __syncthreads();
  }
}

template<typename T, typename U, typename V=T> __global__
void cuApplyLayerNorm(
  V* __restrict__ output_vals,
  U* __restrict__ mean,
  U* __restrict__ invvar,
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const U epsilon,
  const V* __restrict__ gamma,
  const V* __restrict__ beta
  )
{
  cuApplyLayerNorm_<T, U, V>(output_vals, mean, invvar, vals, n1, n2, epsilon, gamma, beta, false);
}

template<typename T, typename U, typename V=T> __global__
void cuApplyRMSNorm(
  V* __restrict__ output_vals,
  U* __restrict__ invvar,
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const U epsilon,
  const V* __restrict__ gamma)
{
  cuApplyLayerNorm_<T, U, V>(output_vals, NULL, invvar, vals, n1, n2, epsilon, gamma, NULL, true);
}


template<typename V> __device__
V clamp_by_magnitude(V curr_gamma, double eps)
{
  const V kMinGamma = V(eps);
  if (curr_gamma >= 0) {
    if (curr_gamma < kMinGamma) {
      return kMinGamma;
    } else {
      return curr_gamma;
    }
  } else {
    if (curr_gamma > -kMinGamma) {
      return -kMinGamma;
    } else {
      return curr_gamma;
    }
  }
}


template<typename T, typename U, typename V, bool MemoryEfficient> __device__
void cuLoadWriteStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf1,
    U* warp_buf2,
    const T* input_or_output,
    const V* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    const V* __restrict__ gamma,
    const V* __restrict__ beta,
    const double eps,
    bool rms_only
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    for (int k = 0;  k < blockDim.y;  ++k) {
      int i2 = i2_off + k;
      int load_idx = i1*n2+i2;
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U c_h = static_cast<U>(input_or_output[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        if (!rms_only) {
          warp_buf1[write_idx] = curr_dout;
          if (MemoryEfficient) {
            U curr_beta = static_cast<U>(beta[i2]);
            warp_buf2[write_idx] = curr_dout * (c_h - curr_beta) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] = curr_dout * (c_h - mean[i1]) * invvar[i1];
          }
        } else {
          if (MemoryEfficient) {
            warp_buf2[write_idx] = curr_dout * (c_h) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] = curr_dout * (c_h) * invvar[i1];
          }
        }
      } else {
        if (!rms_only) {
          warp_buf1[write_idx] = U(0);
        }
        warp_buf2[write_idx] = U(0);
      }
    }
  } else {
    for (int k = 0;  k < blockDim.y;  ++k) {
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (!rms_only) {
        warp_buf1[write_idx] = U(0);
      }
      warp_buf2[write_idx] = U(0);
    }
  }
}

template<typename T, typename U, typename V, bool MemoryEfficient> __device__
void cuLoadAddStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf1,
    U* warp_buf2,
    const T* input_or_output,
    const V* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    const V* __restrict__ gamma,
    const V* __restrict__ beta,
    const double eps,
    bool rms_only
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    for (int k = 0;  k < blockDim.y;  ++k) {
      int i2 = i2_off + k;
      int load_idx = i1*n2+i2;
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U c_h = static_cast<U>(input_or_output[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        if (!rms_only) {
          U curr_beta = static_cast<U>(beta[i2]);
          warp_buf1[write_idx] += curr_dout;
          if (MemoryEfficient) {
            warp_buf2[write_idx] += curr_dout * (c_h - curr_beta) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] += curr_dout * (c_h - mean[i1]) * invvar[i1];
          }
        } else {
          if (MemoryEfficient) {
            warp_buf2[write_idx] += curr_dout * (c_h) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] += curr_dout * (c_h) * invvar[i1];
          }
        }
      }
    }
  }
}


template<typename T, typename U, typename V, bool MemoryEfficient> __global__
void cuComputePartGradGammaBeta(
    const V* __restrict__ dout,
    const T* __restrict__ input_or_output,
    const int n1,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    U epsilon,
    const V* __restrict__ gamma,
    const V* __restrict__ beta,
    U* part_grad_gamma,
    U* part_grad_beta,
    const double eps,
    bool rms_only)
{
    const int numsegs_n1 = (n1+blockDim.y*blockDim.y-1) / (blockDim.y*blockDim.y);
    const int segs_per_block = (numsegs_n1 + gridDim.y - 1) / gridDim.y;
    const int i1_beg = blockIdx.y * segs_per_block * blockDim.y*blockDim.y;
    const int i1_beg_plus_one = (blockIdx.y+1) * segs_per_block * blockDim.y*blockDim.y;
    const int i1_end = i1_beg_plus_one < n1 ? i1_beg_plus_one : n1;
    const int row_stride = blockDim.x+1;
    const int thr_load_col_off = (threadIdx.x*blockDim.y)&(blockDim.x-1);
    const int thr_load_row_off = (threadIdx.x*blockDim.y)/blockDim.x + threadIdx.y*blockDim.y;
    const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;
    SharedMemory<U> shared;
    U* buf = shared.getPointer(); // buf has at least blockDim.x * blockDim.y * blockDim.y + (blockDim.y - 1)*(blockDim.x/blockDim.y) elements
    U* warp_buf1 = (U*)buf;
    U* warp_buf2 = warp_buf1 + blockDim.y * blockDim.y * row_stride;
    // compute partial sums from strided inputs
    // do this to increase number of loads in flight
    cuLoadWriteStridedInputs<T, U, V, MemoryEfficient>(i1_beg,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input_or_output,dout,i1_end,n2,mean,invvar,gamma,beta,eps, rms_only);
    for (int i1_block = i1_beg+blockDim.y*blockDim.y;  i1_block < i1_end;  i1_block+=blockDim.y*blockDim.y) {
      cuLoadAddStridedInputs<T, U, V, MemoryEfficient>(i1_block,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input_or_output,dout,i1_end,n2,mean,invvar,gamma,beta,eps, rms_only);
    }
    __syncthreads();
    // inter-warp reductions
    // sum within each warp
    U acc1 = U(0);
    U acc2 = U(0);
    for (int k = 0;  k < blockDim.y;  ++k) {
      int row1 = threadIdx.y + k*blockDim.y;
      int idx1 = row1*row_stride + threadIdx.x;
      if (!rms_only) {
        acc1 += warp_buf1[idx1];
      }
      acc2 += warp_buf2[idx1];
    }
    if (!rms_only) {
      warp_buf1[threadIdx.y*row_stride+threadIdx.x] = acc1;
    }
    warp_buf2[threadIdx.y*row_stride+threadIdx.x] = acc2;
    __syncthreads();
    // sum all warps
    for (int offset = blockDim.y/2;  offset > 1;  offset /= 2) {
      if (threadIdx.y < offset) {
        int row1 = threadIdx.y;
        int row2 = threadIdx.y + offset;
        int idx1 = row1*row_stride + threadIdx.x;
        int idx2 = row2*row_stride + threadIdx.x;
        if (!rms_only) {
          warp_buf1[idx1] += warp_buf1[idx2];
        }
        warp_buf2[idx1] += warp_buf2[idx2];
      }
      __syncthreads();
    }
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.y == 0 && i2 < n2) {
      int row1 = threadIdx.y;
      int row2 = threadIdx.y + 1;
      int idx1 = row1*row_stride + threadIdx.x;
      int idx2 = row2*row_stride + threadIdx.x;
      if (!rms_only) {
        part_grad_beta[blockIdx.y*n2+i2] = warp_buf1[idx1] + warp_buf1[idx2];
      }
      part_grad_gamma[blockIdx.y*n2+i2] = warp_buf2[idx1] + warp_buf2[idx2];
    }
}

template<typename U, typename V> __global__
void cuComputeGradGammaBeta(
    const U* part_grad_gamma,
    const U* part_grad_beta,
    const int part_size,
    const int n1,
    const int n2,
    V* grad_gamma,
    V* grad_beta,
    bool rms_only)
{
    // sum partial gradients for gamma and beta
    SharedMemory<U> shared;
    U* buf = shared.getPointer();
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (i2 < n2) {
      // each warp does sequential reductions until reduced part_size is num_warps
      int num_warp_reductions = part_size / blockDim.y;
      U sum_gamma = U(0);
      U sum_beta = U(0);
      const U* part_grad_gamma_ptr = part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
      const U* part_grad_beta_ptr = part_grad_beta + threadIdx.y * num_warp_reductions * n2 + i2;
      for (int warp_offset = 0;  warp_offset < num_warp_reductions;  ++warp_offset) {
        sum_gamma += part_grad_gamma_ptr[warp_offset*n2];
        if (!rms_only) {
          sum_beta += part_grad_beta_ptr[warp_offset*n2];
        }
      }
      // inter-warp reductions
      const int nbsize3 = blockDim.x * blockDim.y / 2;
      for (int offset = blockDim.y/2;  offset >= 1;  offset /= 2) {
        // top half write to shared memory
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          buf[write_idx] = sum_gamma;
          if (!rms_only) {
            buf[write_idx+nbsize3] = sum_beta;
          }
        }
        __syncthreads();
        // bottom half sums
        if (threadIdx.y < offset) {
          const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
          sum_gamma += buf[read_idx];
          if (!rms_only) {
            sum_beta += buf[read_idx+nbsize3];
          }
        }
        __syncthreads();
      }
      // write out fully summed gradients
      if (threadIdx.y == 0) {
        grad_gamma[i2] = sum_gamma;
        if (!rms_only) {
          grad_beta[i2] = sum_beta;
        }
      }
    }
}


template<typename T, typename U, typename V, bool MemoryEfficient> __global__
void cuComputeGradInput(
    const V* __restrict__ dout,
    const T* __restrict__ input_or_output,
    const int n1,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    U epsilon,
    const V* gamma,
    const V* beta,
    T* grad_input,
    const double eps,
    bool rms_only)
{
  for (auto i1=blockIdx.y; i1 < n1; i1 += gridDim.y) {
    U sum_loss1 = U(0);
    U sum_loss2 = U(0);
    const T* k_h = input_or_output + i1*n2;
    const V* k_dout = dout + i1*n2;
    const U c_invvar = invvar[i1];
    const U c_mean = !MemoryEfficient ? mean[i1] : 0.;
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL) {
      int l = 4*thrx;
      for (;  l+3 < n2;  l+=4*numx) {
        for (int k = 0;  k < 4;  ++k) {
          const U c_h = static_cast<U>(k_h[l+k]);
          const U c_loss = static_cast<U>(k_dout[l+k]);
          if (!rms_only) {
            sum_loss1 += c_loss * gamma[l+k];
            if (MemoryEfficient) {
              sum_loss2 += c_loss * (c_h - beta[l+k]);
            } else {
              sum_loss2 += c_loss * gamma[l+k] * (c_h - c_mean) * c_invvar;
            }
          } else {
            if (MemoryEfficient) {
              sum_loss2 += c_loss * c_h;
            } else {
              sum_loss2 += c_loss * gamma[l+k] * (c_h) * c_invvar;
            }
          }
        }
      }
      for (;  l < n2;  ++l) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        if (!rms_only) {
          sum_loss1 += c_loss * gamma[l];
          if (MemoryEfficient) {
            sum_loss2 += c_loss * (c_h - beta[l]);
          } else {
            sum_loss2 += c_loss * gamma[l] * (c_h - c_mean) * c_invvar;
          }
        } else {
          if (MemoryEfficient) {
            sum_loss2 += c_loss * c_h;
          } else {
            sum_loss2 += c_loss * gamma[l] * (c_h) * c_invvar;
          }
        }
      }
    } else {
      int l = 4*thrx;
      for (;  l+3 < n2;  l+=4*numx) {
        for (int k = 0;  k < 4;  ++k) {
          const U c_h = static_cast<U>(k_h[l+k]);
          const U c_loss = static_cast<U>(k_dout[l+k]);
          if (!rms_only) {
            sum_loss1 += c_loss;
            if (MemoryEfficient) {
              sum_loss2 += c_loss * c_h;
            } else {
              sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
            }
          } else {
            if (MemoryEfficient) {
              sum_loss2 += c_loss * c_h;
            } else {
              sum_loss2 += c_loss * (c_h) * c_invvar;
            }
          }
        }
      }
      for (;  l < n2;  ++l) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        if (!rms_only) {
          sum_loss1 += c_loss;
          if (MemoryEfficient) {
            sum_loss2 += c_loss * c_h;
          } else {
            sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
          }
        } else {
          if (MemoryEfficient) {
            sum_loss2 += c_loss * c_h;
          } else {
            sum_loss2 += c_loss * (c_h) * c_invvar;
          }
        }
      }
    }
    // intra-warp reductions
    for (int mask = blockDim.x/2;  mask > 0;  mask /= 2) {
      if (!rms_only) {
        sum_loss1 += WARP_SHFL_XOR(sum_loss1, mask);
      }
      sum_loss2 += WARP_SHFL_XOR(sum_loss2, mask);
    }
    // inter-warp reductions
    if (blockDim.y > 1) {
      SharedMemory<U> shared;
      U* buf = shared.getPointer();
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_i = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          if (!rms_only) {
            buf[2*wrt_i] = sum_loss1;
          }
          buf[2*wrt_i+1] = sum_loss2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.y < offset) {
          const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
          if (!rms_only) {
            sum_loss1 += buf[2*read_i];
          }
          sum_loss2 += buf[2*read_i+1];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        if (!rms_only) {
          buf[2*threadIdx.x] = sum_loss1;
        }
        buf[2*threadIdx.x+1] = sum_loss2;
      }
      __syncthreads();
      if (threadIdx.y !=0) {
        if (!rms_only) {
          sum_loss1 = buf[2*threadIdx.x];
        }
        sum_loss2 = buf[2*threadIdx.x+1];
      }
    }
    // all threads now have the two sums over l
    U fH = (U)n2;
    U term1 = (U(1) / fH) * c_invvar;
    T* k_grad_input = grad_input + i1*n2;
    if (gamma != NULL) {
      for (int l = thrx;  l < n2;  l+=numx) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        const U k_gamma = static_cast<U>(clamp_by_magnitude(gamma[l], eps));
        U f_grad_input = fH * c_loss * k_gamma;
        if (!rms_only) {
          const U k_beta = beta[l];
          f_grad_input -= sum_loss1;
          if (MemoryEfficient) {
            f_grad_input -= (c_h - k_beta) / k_gamma * sum_loss2;
          } else {
            f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
          }
        } else {
          if (MemoryEfficient) {
            f_grad_input -= c_h / k_gamma * sum_loss2;
          } else {
            f_grad_input -= c_h * c_invvar * sum_loss2;
          }
        }
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    } else {
      for (int l = thrx;  l < n2;  l+=numx) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss;
        if (!rms_only) {
          f_grad_input -= sum_loss1;
          if (MemoryEfficient) {
            f_grad_input -= c_h * sum_loss2;
          } else {
            f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
          }
        } else {
          if (MemoryEfficient) {
            f_grad_input -= c_h * sum_loss2;
          } else {
            f_grad_input -= c_h * c_invvar * sum_loss2;
          }
        }
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    }
    // prevent race where buf is written again before reads are done
    __syncthreads();
  }
}

void FusedLayerNormCuda(const NDArray& in_arr, const NDArray& ln_scale,
                        const NDArray& ln_bias, NDArray& mean_arr, NDArray& var_arr,
                        NDArray& out_arr, int64_t reduce_dims, 
                        float eps, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(in_arr);
  HT_ASSERT_SAME_DEVICE(in_arr, ln_scale);
  HT_ASSERT_SAME_DEVICE(in_arr, ln_bias);
  HT_ASSERT_SAME_DEVICE(in_arr, mean_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, var_arr); 
  HT_ASSERT_SAME_DEVICE(in_arr, out_arr);

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());

  int ndim = in_arr->ndim();
  int base_dim = 1, last_dim = 1;
  for (int i = 0; i < ndim - reduce_dims; ++i)
    base_dim *= in_arr->shape(i);
  for (int i = ndim - reduce_dims; i < ndim; ++i)
    last_dim *= in_arr->shape(i);
  const dim3 threads(32,4,1);
  auto dprops = Device::dprop(in_arr->device().index());
  const uint64_t maxGridY = dprops.maxGridSize[1];
  size_t n1 = in_arr->numel() / last_dim, n2 = last_dim;
  const dim3 blocks(1, std::min((uint64_t)n1, maxGridY), 1);
  HT_DISPATCH_FLOATING_TYPES(
    in_arr->dtype(), spec_t, "CalculateGradCuda", [&]() {
      int nshared = threads.y > 1 ?
          threads.y * sizeof(float) + (threads.y / 2) * sizeof(float) : 0;
      cuApplyLayerNorm<<<blocks, threads, nshared, hip_stream>>>(
        out_arr->data_ptr<spec_t>(), mean_arr->data_ptr<float>(), 
        var_arr->data_ptr<float>(), in_arr->data_ptr<spec_t>(), 
        n1, n2, float(eps), ln_scale->data_ptr<spec_t>(), 
        ln_bias->data_ptr<spec_t>()); 
  });
  CudaStreamSynchronize(hip_stream);
  NDArray::MarkUsedBy({in_arr, ln_scale, ln_bias, mean_arr, var_arr, out_arr}, stream);
}

void FusedLayerNormGradientCuda(const NDArray& out_grads, const NDArray& in_arr,
                                const NDArray& ln_scale, const NDArray& ln_bias, NDArray& grad_arr,
                                NDArray& grad_scale, NDArray& grad_bias,
                                const NDArray& mean_arr, const NDArray& var_arr,
                                int64_t reduce_dims, float eps, bool inplace, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(out_grads);
  HT_ASSERT_SAME_DEVICE(out_grads, ln_scale);
  HT_ASSERT_SAME_DEVICE(out_grads, in_arr);
  HT_ASSERT_SAME_DEVICE(out_grads, mean_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, var_arr); 
  HT_ASSERT_SAME_DEVICE(out_grads, grad_scale);
  HT_ASSERT_SAME_DEVICE(out_grads, grad_arr);
  HT_ASSERT_SAME_DEVICE(out_grads, grad_bias);

  CUDAStream hip_stream(stream);
  hydraulis::cuda::CUDADeviceGuard guard(hip_stream.device_id());
  hipdnnHandle_t handle = hydraulis::impl::GetCudnnHandle(hip_stream.device_id());

  int ndim = out_grads->ndim();
  size_t total_elements = 1;

  int last_2dim = in_arr->shape(ndim - 1) * in_arr->shape(ndim - 2);

  HTAxes reduce_axes_before = {}, reduce_axes_after = {};
  for (int i = 0; i < ndim; ++i) {
    if (i < ndim - reduce_dims)
      reduce_axes_before.emplace_back(i);
    else
      reduce_axes_after.emplace_back(i);
  }

  for (int i = 0; i < ndim; ++i)
    total_elements *= out_grads->shape(i);
  int lastdim = 1;
  for (size_t i = 0; i < reduce_dims; ++i) {
    lastdim *= out_grads->shape(ndim - 1 - i);
  }

  size_t size = total_elements;
  if (size == 0)
    return;
  size_t n1 = in_arr->numel() / lastdim, n2 = lastdim;
  HT_DISPATCH_FLOATING_TYPES(
      in_arr->dtype(), spec_t, "CalculateGradCuda", [&]() {
      const int part_size = 16;
      const dim3 threads2(32,4,1);
      const dim3 blocks2((n2+threads2.x-1)/threads2.x,part_size,1);
      const int nshared2_a = 2 * sizeof(float) * threads2.y * threads2.y * (threads2.x + 1);
      const int nshared2_b = threads2.x * threads2.y * sizeof(float);
      const int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;
      const auto part_grad_dtype =
        (in_arr->dtype() == DataType::FLOAT16 || in_arr->dtype() == DataType::BFLOAT16) ?
        DataType::FLOAT32 : in_arr->dtype();
      HTShape part_shape = {int64_t(part_size), int64_t(n2)};
      auto part_grad_gamma = NDArray::empty(part_shape, in_arr->device(), part_grad_dtype, stream.stream_index());
      auto part_grad_beta = NDArray::empty_like(part_grad_gamma, stream.stream_index());
      if (inplace) { 
        cuComputePartGradGammaBeta<spec_t, float, spec_t, true><<<blocks2, threads2, nshared2, hip_stream>>>(
                        out_grads->data_ptr<spec_t>(),
                        in_arr->data_ptr<spec_t>(),
                        n1,n2,
                        mean_arr->data_ptr<float>(),
                        var_arr->data_ptr<float>(),
                        float(eps),
                        ln_scale->data_ptr<spec_t>(),
                        ln_bias->data_ptr<spec_t>(),
                        part_grad_gamma->data_ptr<float>(),
                        part_grad_beta->data_ptr<float>(),
                        double(eps),
                        false);
      }
      else {
        cuComputePartGradGammaBeta<spec_t, float, spec_t, false><<<blocks2, threads2, nshared2, hip_stream>>>(
                        out_grads->data_ptr<spec_t>(),
                        in_arr->data_ptr<spec_t>(),
                        n1,n2,
                        mean_arr->data_ptr<float>(),
                        var_arr->data_ptr<float>(),
                        float(eps),
                        ln_scale->data_ptr<spec_t>(),
                        ln_bias->data_ptr<spec_t>(),
                        part_grad_gamma->data_ptr<float>(),
                        part_grad_beta->data_ptr<float>(),
                        double(eps),
                        false);
      }

        const dim3 threads3(32,8,1);
        const dim3 blocks3((n2+threads2.x-1)/threads2.x,1,1);
        const int nshared3 = threads3.x * threads3.y * sizeof(float);
          cuComputeGradGammaBeta<<<blocks3, threads3, nshared3, hip_stream>>>(
                          part_grad_gamma->data_ptr<float>(),
                          part_grad_beta->data_ptr<float>(),
                          part_size,
                          n1,n2,
                          grad_scale->data_ptr<spec_t>(),
                          grad_bias->data_ptr<spec_t>(),
                          false);

        auto dprops = Device::dprop(in_arr->device().index());
        const uint64_t maxGridY = dprops.maxGridSize[1];
        const dim3 blocks1(1, std::min((uint64_t)n1, maxGridY), 1);
        // const dim3 blocks1(1, (uint64_t)n1, 1);
        const dim3 threads1(32,4,1);
        int nshared =
                threads1.y > 1 ?
                threads1.y * threads1.x * sizeof(float) : 0;
        if (inplace) {
          cuComputeGradInput<spec_t, float, spec_t, true><<<blocks1, threads1, nshared, hip_stream>>>(
                  out_grads->data_ptr<spec_t>(),
                  in_arr->data_ptr<spec_t>(),
                  n1,n2,
                  mean_arr->data_ptr<float>(),
                  var_arr->data_ptr<float>(),
                  float(eps),
                  ln_scale->data_ptr<spec_t>(),
                  ln_bias->data_ptr<spec_t>(),
                  grad_arr->data_ptr<spec_t>(),
                  eps,
                  false);
        }
        else {
          cuComputeGradInput<spec_t, float, spec_t, false><<<blocks1, threads1, nshared, hip_stream>>>(
                  out_grads->data_ptr<spec_t>(),
                  in_arr->data_ptr<spec_t>(),
                  n1,n2,
                  mean_arr->data_ptr<float>(),
                  var_arr->data_ptr<float>(),
                  float(eps),
                  ln_scale->data_ptr<spec_t>(),
                  ln_bias->data_ptr<spec_t>(),
                  grad_arr->data_ptr<spec_t>(),
                  eps,
                  false);
        }
      });
  NDArray::MarkUsedBy({out_grads, in_arr, ln_scale, ln_bias, grad_arr,
                       grad_scale, grad_bias, mean_arr, var_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
