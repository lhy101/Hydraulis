#include "hydraulis/core/ndarray.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"
#include "hydraulis/impl/utils/offset_calculator.cuh"
#include "hydraulis/impl/kernel/Vectorized.cuh"

namespace hydraulis {
namespace impl {

void ReluCuda(const NDArray& input, NDArray& output, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output);
  HT_ASSERT_SAME_SHAPE(input, output);

  size_t size = output->numel();
  if (size == 0)
    return;
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ReluCuda", [&]() {
      launch_loop_kernel<spec_t, spec_t>(input, output, size, stream,
                                         [] __device__ (spec_t x) -> spec_t {
                                           spec_t zero = 0;
                                           return (double(x) <= 0) ? zero : x;
                                         });
    });
  NDArray::MarkUsedBy({input, output}, stream);
}

void ReluGradientCuda(const NDArray& input, const NDArray& output_grad,
                      NDArray& input_grad, const Stream& stream) {
  HT_ASSERT_CUDA_DEVICE(input);
  HT_ASSERT_SAME_DEVICE(input, output_grad);
  HT_ASSERT_SAME_DEVICE(input, input_grad);
  HT_ASSERT_SAME_SHAPE(input, output_grad);
  HT_ASSERT_SAME_SHAPE(input, input_grad);

  size_t size = input_grad->numel();
  if (size == 0)
    return;
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "ReluGradientCuda", [&]() {
      launch_loop_kernel<spec_t, spec_t, spec_t>(input, output_grad, input_grad, size, stream,
                                                 [] __device__ (spec_t x, spec_t y) -> spec_t {
                                                   spec_t zero = 0;
                                                   return (double(x) <= 0) ? zero : y;
                                                });
  });
  NDArray::MarkUsedBy({input, output_grad, input_grad}, stream);
}

} // namespace impl
} // namespace hydraulis
