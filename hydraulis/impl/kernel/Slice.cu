#include "hip/hip_runtime.h"
#include "hydraulis/core/ndarray.h"
#include "hydraulis/core/memory_pool.h"
#include "hydraulis/impl/stream/CUDAStream.h"
#include "hydraulis/impl/utils/common_utils.h"
#include "hydraulis/impl/utils/cuda_utils.h"

namespace hydraulis {
namespace impl {

// Out-of-place version of slice and its gradient
/* It is replaced with in-place version. */
template <typename spec_t>
__global__ void slice_kernel(const spec_t* input, spec_t* output,
                             const int64_t* output_shape,
                             const int64_t* input_shape,
                             const int64_t* begin_pos, 
                             size_t ndim,
                             size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  size_t tmp_index = idx;
  size_t i_index = 0;
  int64_t i_mat = 1;

  for (int i = ndim - 1; i >= 0; --i) {
    int64_t offset = begin_pos[i] + tmp_index % output_shape[i];
    tmp_index /= output_shape[i];
    i_index += offset * i_mat;
    i_mat *= input_shape[i];
  }
  output[idx] = input[i_index];
}

template <typename spec_t>
__global__ void slice_gradient_kernel(const spec_t* input, spec_t* output,
                                      const int64_t* output_shape,
                                      const int64_t* input_shape,
                                      const int64_t* begin_pos,
                                      size_t ndim, size_t size) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= size)
    return;
  output[idx] = 0;
  size_t tmp_index = idx;
  size_t i_index = 0;
  int64_t i_mat = 1;
  for (int i = ndim - 1; i >= 0; --i) {
    int64_t offset = tmp_index % output_shape[i];
    if (offset < begin_pos[i] || offset >= begin_pos[i] + input_shape[i])
      return;
    tmp_index /= output_shape[i];
    i_index += (offset - begin_pos[i]) * i_mat;
    i_mat *= input_shape[i];
  }
  output[idx] = input[i_index];
}

void SliceCuda(const NDArray& input, NDArray& output, const HTShape& begin_pos,
               const Stream& stream) {
  HT_ASSERT(input->is_cuda()) << "Input is not on a host device.";
  HT_ASSERT(output->is_cuda()) << "Output is not on a host device.";
  HT_ASSERT(input->device() == output->device())
    << "input and output are not on the same host device. "
    << "Devices: (input) " << input->device() << " vs. (output) "
    << output->device();
  HT_ASSERT(input->ndim() == output->ndim())
    << "input and output has different dims. "
    << "input shape: " << input->shape()
    << ", output shape: " << output->shape();
  size_t ndim = input->ndim();
  size_t o_size = 1;
  for (int i = 0; i < ndim; ++i) {
    HT_ASSERT(begin_pos[i] >= 0);
    HT_ASSERT(begin_pos[i] + output->shape(i) <= input->shape(i))
    << begin_pos[i] << "," << output->shape(i) << "," << input->shape(i);
    o_size *= output->shape(i);
  }
  
  size_t size = o_size;
  if (size == 0)
    return;
  
  auto device_id = input->device().index();
  hydraulis::cuda::CUDADeviceGuard guard(device_id);
  CUDAStream hip_stream(stream);
  auto pos_arr = hydraulis::cuda::to_int64_ndarray(begin_pos, device_id);
  auto i_shape_arr = hydraulis::cuda::to_int64_ndarray(input->shape(), device_id);
  auto o_shape_arr = hydraulis::cuda::to_int64_ndarray(output->shape(), device_id);
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    input->dtype(), spec_t, "SliceCuda", [&]() {
      slice_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        input->data_ptr<spec_t>(), output->data_ptr<spec_t>(),
        o_shape_arr->data_ptr<int64_t>(), 
        i_shape_arr->data_ptr<int64_t>(), 
        pos_arr->data_ptr<int64_t>(), 
        ndim, size);
    });
  NDArray::MarkUsedBy({input, output, pos_arr, i_shape_arr, o_shape_arr}, stream);
}

void SliceGradientCuda(const NDArray& output_grad, NDArray& input_grad,
                       const HTShape& begin_pos, const Stream& stream) {
  HT_ASSERT(output_grad->is_cuda()) << "Output_grad is not on a host device.";
  HT_ASSERT(input_grad->is_cuda()) << "Input_grad is not on a host device.";
  HT_ASSERT(input_grad->device() == output_grad->device())
    << "input_grad and output_grad are not on the same host device. "
    << "Devices: (input_grad) " << input_grad->device() << " vs. (output_grad) "
    << output_grad->device();
  HT_ASSERT(input_grad->ndim() == output_grad->ndim())
    << "input and output grad has different dims. ";
  size_t ndim = output_grad->ndim();
  size_t o_size = 1;
  for (int i = 0; i < ndim; ++i) {
    HT_ASSERT(begin_pos[i] >= 0);
    HT_ASSERT(begin_pos[i] + output_grad->shape(i) <= input_grad->shape(i));
    o_size *= input_grad->shape(i);
  }

  size_t size = input_grad->numel();
  if (size == 0)
    return;
  
  auto device_id = output_grad->device().index();
  hydraulis::cuda::CUDADeviceGuard guard(device_id);
  CUDAStream hip_stream(stream);
  auto pos_arr = hydraulis::cuda::to_int64_ndarray(begin_pos, device_id);
  auto i_shape_arr =
    hydraulis::cuda::to_int64_ndarray(output_grad->shape(), device_id);
  auto o_shape_arr =
    hydraulis::cuda::to_int64_ndarray(input_grad->shape(), device_id);
  dim3 blocks, threads;
  threads.x = MIN(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  blocks.x = DIVUP(size, HT_DEFAULT_NUM_THREADS_PER_BLOCK);
  HT_DISPATCH_INTEGER_AND_FLOATING_TYPES(
    output_grad->dtype(), spec_t, "SliceGradientCuda", [&]() {
      slice_gradient_kernel<spec_t><<<blocks, threads, 0, hip_stream>>>(
        output_grad->data_ptr<spec_t>(), input_grad->data_ptr<spec_t>(),
        o_shape_arr->data_ptr<int64_t>(), 
        i_shape_arr->data_ptr<int64_t>(), 
        pos_arr->data_ptr<int64_t>(), 
        ndim, size);
    });
  NDArray::MarkUsedBy({output_grad, input_grad, pos_arr, i_shape_arr, o_shape_arr}, stream);
}

} // namespace impl
} // namespace hydraulis
